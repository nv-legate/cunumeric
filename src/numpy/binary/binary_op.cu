#include "hip/hip_runtime.h"
/* Copyright 2021 NVIDIA Corporation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */

#include "numpy/binary/binary_op.h"
#include "numpy/binary/binary_op_template.inl"

#include "numpy/cuda_help.h"

namespace legate {
namespace numpy {

using namespace Legion;

template <typename Function, typename RES, typename ARG>
static __global__ void __launch_bounds__(THREADS_PER_BLOCK, MIN_CTAS_PER_SM)
  dense_kernel(size_t volume, Function func, RES* out, const ARG* in1, const ARG* in2)
{
  const size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= volume) return;
  out[idx] = func(in1[idx], in2[idx]);
}

template <typename Function, typename WriteAcc, typename ReadAcc, typename Pitches, typename Rect>
static __global__ void __launch_bounds__(THREADS_PER_BLOCK, MIN_CTAS_PER_SM) generic_kernel(
  size_t volume, Function func, WriteAcc out, ReadAcc in1, ReadAcc in2, Pitches pitches, Rect rect)
{
  const size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= volume) return;
  auto point = pitches.unflatten(idx, rect.lo);
  out[point] = func(in1[point], in2[point]);
}

template <BinaryOpCode OP_CODE, LegateTypeCode CODE, int DIM>
struct BinaryOpImplBody<VariantKind::GPU, OP_CODE, CODE, DIM> {
  using OP  = BinaryOp<OP_CODE, CODE>;
  using ARG = legate_type_of<CODE>;
  using RES = std::result_of_t<OP(ARG, ARG)>;

  void operator()(OP func,
                  AccessorWO<RES, DIM> out,
                  AccessorRO<ARG, DIM> in1,
                  AccessorRO<ARG, DIM> in2,
                  const Pitches<DIM - 1>& pitches,
                  const Rect<DIM>& rect,
                  bool dense) const
  {
    size_t volume       = rect.volume();
    const size_t blocks = (volume + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    if (dense) {
      auto outptr = out.ptr(rect);
      auto in1ptr = in1.ptr(rect);
      auto in2ptr = in2.ptr(rect);
      dense_kernel<<<blocks, THREADS_PER_BLOCK>>>(volume, func, outptr, in1ptr, in2ptr);
    } else {
      generic_kernel<<<blocks, THREADS_PER_BLOCK>>>(volume, func, out, in1, in2, pitches, rect);
    }
  }
};

/*static*/ void BinaryOpTask::gpu_variant(TaskContext& context)
{
  binary_op_template<VariantKind::GPU>(context);
}

}  // namespace numpy
}  // namespace legate
