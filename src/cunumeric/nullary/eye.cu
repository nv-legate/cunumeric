#include "hip/hip_runtime.h"
/* Copyright 2021-2022 NVIDIA Corporation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */

#include "cunumeric/nullary/eye.h"
#include "cunumeric/nullary/eye_template.inl"
#include "cunumeric/cuda_help.h"

namespace cunumeric {

using namespace Legion;

template <typename VAL>
__global__ static void __launch_bounds__(THREADS_PER_BLOCK, MIN_CTAS_PER_SM)
  eye_kernel(const AccessorWO<VAL, 2> out, const Point<2> start, const size_t max)
{
  const size_t offset = blockIdx.x * blockDim.x + threadIdx.x;
  if (offset >= max) return;
  out[start[0] + offset][start[1] + offset] = 1;
}

template <typename VAL>
struct EyeImplBody<VariantKind::GPU, VAL> {
  void operator()(const AccessorWO<VAL, 2>& out,
                  const Point<2>& start,
                  const coord_t distance) const
  {
    const size_t blocks = (distance + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    auto stream         = get_cached_stream();
    eye_kernel<VAL><<<blocks, THREADS_PER_BLOCK, 0, stream>>>(out, start, distance);
    CHECK_CUDA_STREAM(stream);
  }
};

/*static*/ void EyeTask::gpu_variant(TaskContext& context)
{
  eye_template<VariantKind::GPU>(context);
}

}  // namespace cunumeric
