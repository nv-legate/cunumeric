#include "hip/hip_runtime.h"
/* Copyright 2021-2022 NVIDIA Corporation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */

#include "cunumeric/transform/flip.h"
#include "cunumeric/transform/flip_template.inl"

#include "cunumeric/cuda_help.h"

namespace cunumeric {

using namespace Legion;
using namespace legate;

template <typename WriteAcc, typename ReadAcc, typename Pitches, typename Rect>
static __global__ void __launch_bounds__(THREADS_PER_BLOCK, MIN_CTAS_PER_SM)
  flip_kernel(const size_t volume,
              WriteAcc out,
              ReadAcc in,
              Pitches pitches,
              Rect rect,
              Buffer<int32_t, 1> axes,
              const uint32_t num_axes)
{
  const size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= volume) return;
  auto p = pitches.unflatten(idx, rect.lo);
  auto q = p;
  for (uint32_t idx = 0; idx < num_axes; ++idx) q[axes[idx]] = rect.hi[axes[idx]] - q[axes[idx]];
  out[p] = in[q];
}

template <LegateTypeCode CODE, int32_t DIM>
struct FlipImplBody<VariantKind::GPU, CODE, DIM> {
  using VAL = legate_type_of<CODE>;

  void operator()(AccessorWO<VAL, DIM> out,
                  AccessorRO<VAL, DIM> in,
                  const Pitches<DIM - 1>& pitches,
                  const Rect<DIM>& rect,
                  legate::Span<const int32_t> axes) const

  {
    const size_t volume = rect.volume();
    const size_t blocks = (volume + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    auto num_axes       = axes.size();
    auto gpu_axes       = create_buffer<int32_t>(num_axes, Memory::Kind::Z_COPY_MEM);
    for (uint32_t idx = 0; idx < num_axes; ++idx) gpu_axes[idx] = axes[idx];
    auto stream = get_cached_stream();
    flip_kernel<<<blocks, THREADS_PER_BLOCK, 0, stream>>>(
      volume, out, in, pitches, rect, gpu_axes, num_axes);
    CHECK_CUDA_STREAM(stream);
  }
};

/*static*/ void FlipTask::gpu_variant(TaskContext& context)
{
  flip_template<VariantKind::GPU>(context);
}

}  // namespace cunumeric
