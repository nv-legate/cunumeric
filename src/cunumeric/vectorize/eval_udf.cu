/* Copyright 2023 NVIDIA Corporation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */

#include "cunumeric/vectorize/eval_udf.h"
#include "cunumeric/cuda_help.h"
#include "cunumeric/pitches.h"
#include <regex>
#include <hip/hip_runtime.h>

namespace cunumeric {

// using namespace Legion;
using namespace legate;

struct EvalUdfGPU {
  template <Type::Code CODE, int DIM>
  void operator()(EvalUdfArgs& args) const
  {
    using VAL = legate_type_of<CODE>;
    Rect<DIM> rect;

    size_t input_size = args.inputs.size();
    hipFunction_t func   = get_udf(args.hash);

    // Filling up the buffer with arguments
    size_t buffer_size = (input_size + args.scalars.size()) * sizeof(void*);
    buffer_size += sizeof(size_t);  // size
    buffer_size += sizeof(size_t);  // dim
    buffer_size += sizeof(void*);   // pitches
    buffer_size += sizeof(void*);   // strides

    std::vector<char> arg_buffer(buffer_size);
    char* raw_arg_buffer = arg_buffer.data();

    auto p = raw_arg_buffer;
    size_t strides[DIM];
    size_t size = 1;
    if (input_size > 0) {
      rect = args.inputs[0].shape<DIM>();
      size = rect.volume();
      for (size_t i = 0; i < input_size; i++) {
        if (i < args.num_outputs) {
          auto out                           = args.outputs[i].write_accessor<VAL, DIM>(rect);
          *reinterpret_cast<const void**>(p) = out.ptr(rect, strides);
        } else {
          auto in                            = args.inputs[i].read_accessor<VAL, DIM>(rect);
          *reinterpret_cast<const void**>(p) = in.ptr(rect, strides);
        }
        p += sizeof(void*);
      }
    }
    for (auto scalar : args.scalars) {
      memcpy(p, scalar.ptr(), scalar.size());
      p += scalar.size();
    }
    memcpy(p, &size, sizeof(size_t));
    size_t dim = DIM;
    p += sizeof(size_t);
    memcpy(p, &dim, sizeof(size_t));
    p += sizeof(size_t);
    Pitches<DIM - 1> pitches;
    size_t volume = pitches.flatten(rect);
    // create buffers for pitches and strides since
    // we need to pass pointer to device memory
    auto device_pitches = create_buffer<int64_t>(Point<1>(DIM - 1), Memory::Kind::Z_COPY_MEM);
    auto device_strides = create_buffer<int64_t>(Point<1>(DIM), Memory::Kind::Z_COPY_MEM);
    for (size_t i = 0; i < DIM; i++) {
      if (i != DIM - 1) { device_pitches[Point<1>(i)] = pitches.data()[i]; }
      device_strides[Point<1>(i)] = strides[i];
    }
    *reinterpret_cast<const void**>(p) = device_pitches.ptr(Point<1>(0));
    p += sizeof(void*);
    *reinterpret_cast<const void**>(p) = device_strides.ptr(Point<1>(0));
    p += sizeof(void*);

    void* config[] = {
      HIP_LAUNCH_PARAM_BUFFER_POINTER,
      static_cast<void*>(raw_arg_buffer),
      HIP_LAUNCH_PARAM_BUFFER_SIZE,
      &buffer_size,
      HIP_LAUNCH_PARAM_END,
    };

    const uint32_t gridDimX = (size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    const uint32_t gridDimY = 1;
    const uint32_t gridDimZ = 1;

    const uint32_t blockDimX = THREADS_PER_BLOCK;
    const uint32_t blockDimY = 1;
    const uint32_t blockDimZ = 1;

    auto stream = get_cached_stream();

    hipError_t status = hipModuleLaunchKernel(
      func, gridDimX, gridDimY, gridDimZ, blockDimX, blockDimY, blockDimZ, 0, stream, NULL, config);
    if (status != hipSuccess) {
      fprintf(stderr, "Failed to launch a CUDA kernel\n");
      assert(false);
    }

    CHECK_CUDA_STREAM(stream);
  }
};

/*static*/ void EvalUdfTask::gpu_variant(TaskContext& context)
{
  uint32_t num_outputs = context.scalars()[0].value<uint32_t>();
  uint32_t num_scalars = context.scalars()[1].value<uint32_t>();
  std::vector<Scalar> scalars;
  for (size_t i = 2; i < (2 + num_scalars); i++) scalars.push_back(context.scalars()[i]);

  int64_t ptx_hash = context.scalars()[2 + num_scalars].value<int64_t>();

  EvalUdfArgs args{0,
                   context.inputs(),
                   context.outputs(),
                   scalars,
                   num_outputs,
                   legate::Processor::get_executing_processor(),
                   ptx_hash};
  size_t dim = 1;
  if (args.inputs.size() > 0) {
    dim = args.inputs[0].dim() == 0 ? 1 : args.inputs[0].dim();
    double_dispatch(dim, args.inputs[0].code(), EvalUdfGPU{}, args);
  } else {
    Type::Code code = Type::Code::BOOL;
    double_dispatch(dim, code, EvalUdfGPU{}, args);
  }
}
}  // namespace cunumeric
