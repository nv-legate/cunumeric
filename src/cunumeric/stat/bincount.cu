#include "hip/hip_runtime.h"
/* Copyright 2021 NVIDIA Corporation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */

#include "cunumeric/stat/bincount.h"
#include "cunumeric/stat/bincount_template.inl"

#include "cunumeric/cuda_help.h"

namespace cunumeric {

using namespace Legion;

template <typename VAL>
static __device__ inline void _bincount(int32_t* bins,
                                        AccessorRO<VAL, 1> rhs,
                                        const size_t volume,
                                        const size_t num_bins,
                                        Point<1> origin)
{
  // Initialize the bins to 0
  for (int32_t bin = threadIdx.x; bin < num_bins; bin += blockDim.x) bins[bin] = 0;
  __syncthreads();

  // Start reading values and do atomic updates to shared
  // Since these are 32 bit counts then we know they are native
  // atomics and willl therefore be "fast"
  size_t offset       = blockIdx.x * blockDim.x + threadIdx.x;
  const size_t stride = gridDim.x * blockDim.x;
  while (offset < volume) {
    const auto x   = origin[0] + offset;
    const auto bin = rhs[x];
    assert(bin < num_bins);
    SumReduction<int32_t>::fold<false>(bins[bin], 1);
    // Now get the next offset
    offset += stride;
  }
  // Wait for everyone to be done
  __syncthreads();
}

template <typename VAL>
static __device__ inline void _weighted_bincount(double* bins,
                                                 AccessorRO<VAL, 1> rhs,
                                                 AccessorRO<double, 1> weights,
                                                 const size_t volume,
                                                 const size_t num_bins,
                                                 Point<1> origin)
{
  // Initialize the bins to 0
  for (int32_t bin = threadIdx.x; bin < num_bins; bin += blockDim.x) bins[bin] = 0;
  __syncthreads();

  // Start reading values and do atomic updates to shared
  // Since these are 32 bit counts then we know they are native
  // atomics and willl therefore be "fast"
  size_t offset       = blockIdx.x * blockDim.x + threadIdx.x;
  const size_t stride = gridDim.x * blockDim.x;
  while (offset < volume) {
    const auto x   = origin[0] + offset;
    const auto bin = rhs[x];
    assert(bin < num_bins);
    SumReduction<double>::fold<false>(bins[bin], weights[x]);
    // Now get the next offset
    offset += stride;
  }
  // Wait for everyone to be done
  __syncthreads();
}

template <typename VAL>
static __global__ void __launch_bounds__(THREADS_PER_BLOCK, MIN_CTAS_PER_SM)
  bincount_kernel_rd(AccessorRD<SumReduction<int64_t>, false, 1> lhs,
                     AccessorRO<VAL, 1> rhs,
                     const size_t volume,
                     const size_t num_bins,
                     Point<1> origin)
{
  extern __shared__ char array[];
  auto bins = reinterpret_cast<int32_t*>(array);
  _bincount(bins, rhs, volume, num_bins, origin);
  // Now do the atomics out to global memory
  for (int32_t bin = threadIdx.x; bin < num_bins; bin += blockDim.x) {
    const auto count = bins[bin];
    if (count > 0) lhs.reduce(bin, count);
  }
}

template <typename VAL>
static __global__ void __launch_bounds__(THREADS_PER_BLOCK, MIN_CTAS_PER_SM)
  bincount_kernel_rw(AccessorRW<int64_t, 1> lhs,
                     AccessorRO<VAL, 1> rhs,
                     const size_t volume,
                     const size_t num_bins,
                     Point<1> origin)
{
  extern __shared__ char array[];
  auto bins = reinterpret_cast<int32_t*>(array);
  _bincount(bins, rhs, volume, num_bins, origin);
  // Now do the atomics out to global memory
  for (int32_t bin = threadIdx.x; bin < num_bins; bin += blockDim.x) {
    const auto count = bins[bin];
    if (count > 0) SumReduction<int64_t>::fold<false>(lhs[bin], count);
  }
}

template <typename VAL>
static __global__ void __launch_bounds__(THREADS_PER_BLOCK, MIN_CTAS_PER_SM)
  weighted_bincount_kernel_rd(AccessorRD<SumReduction<double>, false, 1> lhs,
                              AccessorRO<VAL, 1> rhs,
                              AccessorRO<double, 1> weights,
                              const size_t volume,
                              const size_t num_bins,
                              Point<1> origin)
{
  extern __shared__ char array[];
  auto bins = reinterpret_cast<double*>(array);
  _weighted_bincount(bins, rhs, weights, volume, num_bins, origin);
  // Now do the atomics out to global memory
  for (int32_t bin = threadIdx.x; bin < num_bins; bin += blockDim.x) {
    const auto weight = bins[bin];
    lhs.reduce(bin, weight);
  }
}

template <typename VAL>
static __global__ void __launch_bounds__(THREADS_PER_BLOCK, MIN_CTAS_PER_SM)
  weighted_bincount_kernel_rw(AccessorRW<double, 1> lhs,
                              AccessorRO<VAL, 1> rhs,
                              AccessorRO<double, 1> weights,
                              const size_t volume,
                              const size_t num_bins,
                              Point<1> origin)
{
  extern __shared__ char array[];
  auto bins = reinterpret_cast<double*>(array);
  _weighted_bincount(bins, rhs, weights, volume, num_bins, origin);
  // Now do the atomics out to global memory
  for (int32_t bin = threadIdx.x; bin < num_bins; bin += blockDim.x) {
    const auto weight = bins[bin];
    SumReduction<double>::fold<false>(lhs[bin], weight);
  }
}

template <LegateTypeCode CODE>
struct BincountImplBody<VariantKind::GPU, CODE> {
  using VAL = legate_type_of<CODE>;

  void operator()(AccessorRD<SumReduction<int64_t>, false, 1> lhs,
                  const AccessorRO<VAL, 1>& rhs,
                  const Rect<1>& rect,
                  const Rect<1>& lhs_rect) const
  {
    const auto volume   = rect.volume();
    const auto num_bins = lhs_rect.volume();
    const auto bin_size = num_bins * sizeof(int32_t);

    int32_t num_ctas = 0;
    hipOccupancyMaxActiveBlocksPerMultiprocessor(
      &num_ctas, bincount_kernel_rd<VAL>, THREADS_PER_BLOCK, bin_size);
    assert(num_ctas > 0);
    // Launch a kernel with this number of CTAs
    bincount_kernel_rd<VAL>
      <<<num_ctas, THREADS_PER_BLOCK, bin_size>>>(lhs, rhs, volume, num_bins, rect.lo);
  }

  void operator()(const AccessorRW<int64_t, 1>& lhs,
                  const AccessorRO<VAL, 1>& rhs,
                  const Rect<1>& rect,
                  const Rect<1>& lhs_rect) const
  {
    const auto volume   = rect.volume();
    const auto num_bins = lhs_rect.volume();
    const auto bin_size = num_bins * sizeof(int32_t);

    int32_t num_ctas = 0;
    hipOccupancyMaxActiveBlocksPerMultiprocessor(
      &num_ctas, bincount_kernel_rw<VAL>, THREADS_PER_BLOCK, bin_size);
    assert(num_ctas > 0);
    // Launch a kernel with this number of CTAs
    bincount_kernel_rw<VAL>
      <<<num_ctas, THREADS_PER_BLOCK, bin_size>>>(lhs, rhs, volume, num_bins, rect.lo);
  }

  void operator()(AccessorRD<SumReduction<double>, false, 1> lhs,
                  const AccessorRO<VAL, 1>& rhs,
                  const AccessorRO<double, 1>& weights,
                  const Rect<1>& rect,
                  const Rect<1>& lhs_rect) const
  {
    const auto volume   = rect.volume();
    const auto num_bins = lhs_rect.volume();
    const auto bin_size = num_bins * sizeof(double);

    int32_t num_ctas = 0;
    hipOccupancyMaxActiveBlocksPerMultiprocessor(
      &num_ctas, weighted_bincount_kernel_rd<VAL>, THREADS_PER_BLOCK, bin_size);
    assert(num_ctas > 0);
    // Launch a kernel with this number of CTAs
    weighted_bincount_kernel_rd<VAL>
      <<<num_ctas, THREADS_PER_BLOCK, bin_size>>>(lhs, rhs, weights, volume, num_bins, rect.lo);
  }

  void operator()(const AccessorRW<double, 1>& lhs,
                  const AccessorRO<VAL, 1>& rhs,
                  const AccessorRO<double, 1>& weights,
                  const Rect<1>& rect,
                  const Rect<1>& lhs_rect) const
  {
    const auto volume   = rect.volume();
    const auto num_bins = lhs_rect.volume();
    const auto bin_size = num_bins * sizeof(double);

    int32_t num_ctas = 0;
    hipOccupancyMaxActiveBlocksPerMultiprocessor(
      &num_ctas, weighted_bincount_kernel_rw<VAL>, THREADS_PER_BLOCK, bin_size);
    assert(num_ctas > 0);
    // Launch a kernel with this number of CTAs
    weighted_bincount_kernel_rw<VAL>
      <<<num_ctas, THREADS_PER_BLOCK, bin_size>>>(lhs, rhs, weights, volume, num_bins, rect.lo);
  }
};

/*static*/ void BincountTask::gpu_variant(TaskContext& context)
{
  bincount_template<VariantKind::GPU>(context);
}

}  // namespace cunumeric
