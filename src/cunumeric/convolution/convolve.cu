#include "hip/hip_runtime.h"
/* Copyright 2021-2022 NVIDIA Corporation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */

#include "cunumeric/divmod.h"
#include "cunumeric/cuda_help.h"
#include "cunumeric/convolution/convolve.h"
#include "cunumeric/convolution/convolve_common.h"
#include "cunumeric/convolution/convolve_template.inl"

#include <hipfft/hipfftXt.h>

namespace cunumeric {

using namespace Legion;
using namespace legate;

////////////////////////////////////
// Direct convolution implementation
////////////////////////////////////

// Convolution should be able to hit FMA throughput limits
// on the GPU due to the amount of FLOPs needed to be performed
// given the amount of data loaded. This is especially true of
// larger convolution filters. In order to hit these limits though
// we need to make sure that the GPU is fed data appropriately.
// We have two different kernels to handle different sized filters.

// Small Tile Case
// In the small tile case, a reasonable tile input including the
// all the boundary values for a given filter tile can fit in the
// shared memory of the SM, allowing the threadblock to fully
// compute an entire tile of output points in a single pass.
// If the tile is small enough, we even try to get multiple CTAs/SM
// in order to better pipeline data loading with compute.

// Large Tile Case
// For inputs where the filter is very large and it is impossible
// to fit a reasonable sized tile into shared memory, we tile both
// the output and the filter and make multiple passes over the data
// to create reasonable sized input tiles that fit in shared memory.
// If possible we also attempt to tile for the L2 cache as well so
// that threadblocks walking through memory together can hopefully
// hit in the L2 more often than not when loading data

template <int DIM>
struct ConvolutionInitArgs {
 public:
  FastDivmodU64 pitches[DIM];
};

template <typename VAL, int DIM>
__global__ static void __launch_bounds__(THREADS_PER_BLOCK, 4)
  convolution_init(const AccessorWO<VAL, DIM> out,
                   const Point<DIM> subrect_lo,
                   const ConvolutionInitArgs<DIM> args,
                   const size_t volume)
{
  size_t offset = blockIdx.x * blockDim.x + threadIdx.x;
  if (offset >= volume) return;
  Point<DIM> point = subrect_lo;
#pragma unroll
  for (int d = 0; d < DIM; d++) point[d] += args.pitches[d].divmod(offset, offset);
  out[point] = VAL{0};
}

// We want to run the convolution kernel with as large a shared memory
// tile as possible to avoid duplicate loading of data and maximize
// compute intensity. Therefore we're always going to run with 1 CTA
// per SM, but we still want enough thread-level parallelism, so we
// set this to the maximum number of warps in a threadblock
// Note that a lot of this code assumes this is a power of 2
#define CONVOLUTION_THREADS 1024
// The idea behind THREAD_OUTPUTS is to figure out how many registers
// we will be willing to assign to hold the partial output accumulations
// in each thread without using too many registers. Every GPU (with one
// exception) has 64K 32-bit registers per SM. We key off that and want
// to allocate a quarter of those registers for holding the partial accumulations
// We assume here that sizeof(VAL) is a power of 2
#define THREAD_OUTPUTS(TYPE) 1
//((4/*bytes/reg*/ * ((65536/8)/CONVOLUTION_THREADS)/*regs/thread*/) / sizeof(TYPE))

template <int DIM, int POINTS>
struct ConvolutionLargeTileArgs {
  FastDivmod l1_output_tile_pitches[DIM];
  FastDivmod l1_input_pitches[DIM];
  FastDivmod l1_filter_pitches[DIM];
  FastDivmod l1_output_pitches[DIM];
  Point<DIM> l2_output_limits;
  Point<DIM, unsigned> point_offsets[POINTS];
  Point<DIM, unsigned> l2_output_tile;
  Point<DIM, unsigned> l2_filter_tile;
  Point<DIM, unsigned> l1_output_tile;
  Point<DIM, unsigned> l1_filter_tile;
  unsigned total_l2_outputs;
  unsigned total_l1_outputs;
  unsigned total_l1_filters;
  unsigned total_l1_points;
  unsigned l1_filter_points;
  unsigned l1_input_points;
  unsigned shared_input_offset;
  unsigned uniform_input_stride;
  unsigned shared_input_bound;
};

template <typename VAL, int DIM, int POINTS>
__global__ static void __launch_bounds__(CONVOLUTION_THREADS, 1)
  convolution_large_tile(const AccessorWO<VAL, DIM> out,
                         const AccessorRO<VAL, DIM> filter,
                         const AccessorRO<VAL, DIM> in,
                         const Rect<DIM> root_rect,
                         const Rect<DIM> subrect,
                         const Rect<DIM> l2_filter_rect,
                         const Point<DIM> l2_input_start,
                         const Point<DIM> l2_input_stop,
                         const Point<DIM> l1_input_start,
                         const Point<DIM, unsigned> zero,
                         const Point<DIM, unsigned> one,
                         const ConvolutionLargeTileArgs<DIM, POINTS> args)
{
  // Deal with compiler shared memory stupidity
  extern __shared__ uint8_t buffer[];
  // Technically this is illegal C++, but there's no other way to do it
  VAL* sharedmem = (VAL*)buffer;
  Point<DIM, unsigned> thread_offset;
  int offset = threadIdx.x;
#pragma unroll
  for (int d = 0; d < DIM; d++) thread_offset[d] = args.l1_output_pitches[d].divmod(offset, offset);
  Point<DIM> l2_output_offset = zero;
  for (unsigned l2_outidx = 0; l2_outidx < args.total_l2_outputs; l2_outidx++) {
    // Do a quick check here to see if all the inputs are contained for this tile
    // l2_input_start = subrect.lo + args.extents - l2_filter_rect.hi - one - centers
    // l2_input_stop = subrect.lo + l2_output_tile - one + args.extents - l2_filter_rect.lo - one -
    // centers
    const Rect<DIM> l2_input_rect(l2_input_start + l2_output_offset,
                                  l2_input_stop + l2_output_offset);
    const bool input_contained = root_rect.contains(l2_input_rect);
    // Iterate the L1 output tiles that this threadblock should compute for the L2 output
    for (unsigned l1_outidx = blockIdx.x; l1_outidx < args.total_l1_outputs;
         l1_outidx += gridDim.x) {
      Point<DIM, unsigned> l1_output_offset;
      offset = l1_outidx;
#pragma unroll
      for (int d = 0; d < DIM; d++)
        l1_output_offset[d] =
          args.l1_output_tile_pitches[d].divmod(offset, offset) * args.l1_output_tile[d];
      // Handle the boundary case where an L1 tile is not contained in the L2 tile
      // becasue the L2 tile is overlapping a boundary. Note this decisions is the
      // same for all the threads in the threadblock so no bad divergence
      bool output_contained = true;
#pragma unroll
      for (int d = 0; d < DIM; d++) {
        if ((subrect.lo[d] + l2_output_offset[d] + l1_output_offset[d]) <= subrect.hi[d]) continue;
        output_contained = false;
        break;
      }
      if (!output_contained) continue;
      // Initialize our point data
      VAL acc[POINTS];
#pragma unroll
      for (int p = 0; p < POINTS; p++) acc[p] = VAL{0};
      // Iterate over the l1 filter tiles
      Point<DIM, unsigned> l1_filter_offset = zero;
      for (unsigned l1_fidx = 0; l1_fidx < args.total_l1_filters; l1_fidx++) {
        // Wait for any previous readers to be done
        __syncthreads();
// Load the filter into shared memory
// Unroll this a few times to get some memory level parallelims
#pragma unroll 4
        for (unsigned fidx = threadIdx.x; fidx < args.l1_filter_points; fidx += blockDim.x) {
          Point<DIM> filter_point = l2_filter_rect.lo + l1_filter_offset;
          offset                  = fidx;
#pragma unroll
          for (int d = 0; d < DIM; d++)
            filter_point[d] += args.l1_filter_pitches[d].divmod(offset, offset);
          if (l2_filter_rect.contains(filter_point))
            sharedmem[fidx] = filter[filter_point];
          else
            sharedmem[fidx] = VAL{0};
        }
        // Load the input into shared memory
        // Compute the input start point
        // input_start = subrect.lo + extents - l2_filter_rect.lo - l1_filter_tile - centers
        Point<DIM> input_start = l1_input_start + l2_output_offset + l1_output_offset;
        input_start -= l1_filter_offset;
// Unroll this a few times to get some memory level parallelism
#pragma unroll 4
        for (unsigned idx = threadIdx.x; idx < args.l1_input_points; idx += blockDim.x) {
          Point<DIM> input_point = input_start;
          offset                 = idx;
#pragma unroll
          for (int d = 0; d < DIM; d++)
            input_point[d] += args.l1_input_pitches[d].divmod(offset, offset);
          if (input_contained || root_rect.contains(input_point))
            sharedmem[args.shared_input_offset + idx] = in[input_point];
          else
            sharedmem[args.shared_input_offset + idx] = VAL{0};
        }
        // Wait for everything to be loaded into shared memory
        __syncthreads();
        // Iterate the points in the filter
        // We can safely iterate all the filter points and input points
        // because we wrote zeros into shared memory for everything that
        // was out of bounds
        Point<DIM, unsigned> filter_point = zero;
        if (args.uniform_input_stride) {
          // Each point is a constant offset in shared from the others
          unsigned input_offset = args.shared_input_offset;
#pragma unroll
          for (int d = 0; d < DIM; d++)
            input_offset +=
              args.l1_input_pitches[d].divisor * (thread_offset[d] + args.l1_filter_tile[d] - 1);
          if (args.shared_input_bound) {
            for (unsigned fidx = 0; fidx < args.l1_filter_points; fidx++) {
              // Use shared memory broadcasting functionality to avoid bank conflicts
              const VAL filter_value = sharedmem[fidx];
              unsigned point_offset  = input_offset;
#pragma unroll
              for (int p = 0; p < POINTS; p++) {
                if (args.shared_input_bound <= point_offset) break;
                acc[p] = acc[p] + filter_value * sharedmem[point_offset];
                point_offset += args.uniform_input_stride;
              }
// Step to the next filter point and update the input stride
#pragma unroll
              for (int d = DIM - 1; d >= 0; d--) {
                filter_point[d]++;
                input_offset -= args.l1_input_pitches[d].divisor;
                if (filter_point[d] == args.l1_filter_tile[d]) {
                  input_offset += args.l1_filter_tile[d] * args.l1_input_pitches[d].divisor;
                  filter_point[d] = 0;
                } else {
                  break;
                }
              }
            }
          } else {
            for (unsigned fidx = 0; fidx < args.l1_filter_points; fidx++) {
              // Use shared memory broadcasting functionality to avoid bank conflicts
              const VAL filter_value = sharedmem[fidx];
              unsigned point_offset  = input_offset;
#pragma unroll
              for (int p = 0; p < POINTS; p++) {
                acc[p] = acc[p] + filter_value * sharedmem[point_offset];
                point_offset += args.uniform_input_stride;
              }
// Step to the next filter point and update the input stride
#pragma unroll
              for (int d = DIM - 1; d >= 0; d--) {
                filter_point[d]++;
                input_offset -= args.l1_input_pitches[d].divisor;
                if (filter_point[d] == args.l1_filter_tile[d]) {
                  input_offset += args.l1_filter_tile[d] * args.l1_input_pitches[d].divisor;
                  filter_point[d] = 0;
                } else {
                  break;
                }
              }
            }
          }
        } else {
          // Need to compute the input offset uniquely for each point
          Point<DIM, unsigned> input_point = thread_offset + args.l1_filter_tile - one;
          unsigned point_offsets[POINTS];
#pragma unroll
          for (int p = 0; p < POINTS; p++) {
            point_offsets[p] = args.shared_input_offset;
#pragma unroll
            for (int d = 0; d < DIM; d++)
              point_offsets[p] +=
                (input_point[d] + args.point_offsets[p][d]) * args.l1_input_pitches[d].divisor;
          }
          unsigned filter_offset = 0;
          if (args.shared_input_bound) {
            for (unsigned fidx = 0; fidx < args.l1_filter_points; fidx++) {
              // Use shared memory broadcasting functionality to avoid bank conflicts
              const VAL filter_value = sharedmem[fidx];
#pragma unroll
              for (int p = 0; p < POINTS; p++) {
                unsigned point_offset = point_offsets[p] - filter_offset;
                if (args.shared_input_bound <= point_offset) continue;
                acc[p] = acc[p] + filter_value * sharedmem[point_offset];
              }
// Step to the next filter point
#pragma unroll
              for (int d = DIM - 1; d >= 0; d--) {
                filter_point[d]++;
                filter_offset += args.l1_input_pitches[d].divisor;
                if (filter_point[d] == args.l1_filter_tile[d]) {
                  filter_offset -= args.l1_filter_tile[d] * args.l1_input_pitches[d].divisor;
                  filter_point[d] = 0;
                } else {
                  break;
                }
              }
            }
          } else {
            for (unsigned fidx = 0; fidx < args.l1_filter_points; fidx++) {
              // Use shared memory broadcasting functionality to avoid bank conflicts
              const VAL filter_value = sharedmem[fidx];
#pragma unroll
              for (int p = 0; p < POINTS; p++) {
                unsigned point_offset = point_offsets[p] - filter_offset;
                acc[p]                = acc[p] + filter_value * sharedmem[point_offset];
              }
// Step to the next filter point
#pragma unroll
              for (int d = DIM - 1; d >= 0; d--) {
                filter_point[d]++;
                filter_offset += args.l1_input_pitches[d].divisor;
                if (filter_point[d] == args.l1_filter_tile[d]) {
                  filter_offset -= args.l1_filter_tile[d] * args.l1_input_pitches[d].divisor;
                  filter_point[d] = 0;
                } else {
                  break;
                }
              }
            }
          }
        }
// Step to the next L1 filter tile
#pragma unroll
        for (int d = DIM - 1; d >= 0; d--) {
          l1_filter_offset[d] += args.l1_filter_tile[d];
          if (args.l2_filter_tile[d] <= l1_filter_offset[d])
            l1_filter_offset[d] = 0;
          else
            break;
        }
      }
      // Now we can stream our accumulators back to the output
      Point<DIM> output = subrect.lo + l2_output_offset + (l1_output_offset + thread_offset);
      if (input_contained) {
        // If the input was contained, then so is the output
        if (args.total_l1_points) {
          unsigned index = threadIdx.x;
#pragma unroll
          for (int p = 0; p < POINTS; p++) {
            if (args.total_l1_points <= index) break;
            VAL* ptr = out.ptr(output + args.point_offsets[p]);
            // Make sure we don't pollute the L2 cache
            VAL value = load_streaming<VAL>(ptr);
            store_streaming<VAL>(ptr, value + acc[p]);
            index += blockDim.x;
          }
        } else {
#pragma unroll
          for (int p = 0; p < POINTS; p++) {
            VAL* ptr = out.ptr(output + args.point_offsets[p]);
            // Make sure we don't pollute the L2 cache
            VAL value = load_streaming<VAL>(ptr);
            store_streaming<VAL>(ptr, value + acc[p]);
          }
        }
      } else {
        // Input was not contained, so the output might not be either, do checks
        if (args.total_l1_points) {
          unsigned index = threadIdx.x;
#pragma unroll
          for (int p = 0; p < POINTS; p++) {
            if (args.total_l1_points <= index) break;
            Point<DIM> point = output + args.point_offsets[p];
            if (!subrect.contains(point)) break;
            VAL* ptr = out.ptr(point);
            // Make sure we don't pollute the L2 cache
            VAL value = load_streaming<VAL>(ptr);
            store_streaming<VAL>(ptr, value + acc[p]);
            index += blockDim.x;
          }
        } else {
#pragma unroll
          for (int p = 0; p < POINTS; p++) {
            Point<DIM> point = output + args.point_offsets[p];
            if (!subrect.contains(point)) continue;
            VAL* ptr = out.ptr(point);
            // Make sure we don't pollute the L2 cache
            VAL value = load_streaming<VAL>(ptr);
            store_streaming<VAL>(ptr, value + acc[p]);
          }
        }
      }
    }
// Step to the next output tile
#pragma unroll
    for (int d = DIM - 1; d >= 0; d--) {
      l2_output_offset[d] += args.l2_output_tile[d];
      if (args.l2_output_limits[d] <= l2_output_offset[d])
        l2_output_offset[d] = 0;
      else
        break;
    }
  }
}

template <int DIM>
struct ConvolutionSmallTileArgs {
  FastDivmodU64 grid_pitches[DIM];
  FastDivmodU64 block_pitches[DIM];
  FastDivmodU64 input_pitches[DIM];
  unsigned block_tiles[DIM];
  unsigned filter_centers[DIM];
  unsigned filter_extents[DIM];
  Point<DIM> delta_lo, delta_hi;
  size_t filter_volume;
  size_t tile_volume;
  size_t input_volume;
};

template <typename VAL, int DIM>
__global__ static void __launch_bounds__(512, 2)
  convolution_small_tile1(const AccessorWO<VAL, DIM> out,
                          const AccessorRO<VAL, DIM> filter,
                          const AccessorRO<VAL, DIM> in,
                          const Rect<DIM> root_rect,
                          const Rect<DIM> subrect,
                          const Rect<DIM> filter_rect,
                          const ConvolutionSmallTileArgs<DIM> args)
{
  // Deal with compiler shared memory stupidity
  extern __shared__ uint8_t buffer[];
  // Technically this illegal C++, but there's no other way to do it
  VAL* input = (VAL*)buffer;
  // Compute the origin point of the block
  size_t offset          = blockIdx.x;
  Point<DIM> block_point = subrect.lo;
#pragma unroll
  for (int d = 0; d < DIM; d++)
    block_point[d] += args.grid_pitches[d].divmod(offset, offset) * args.block_tiles[d];
  // Load in the shared memory for this block
  Point<DIM> tile_point;
  const Rect<DIM> input_bounds(block_point - args.delta_lo, block_point + args.delta_hi);
  const bool input_contained = root_rect.contains(input_bounds);
  if (input_contained) {
// All the points are contained, so no need for point-wise tests
// Unroll this four times to try to pipeline loads
#pragma unroll 4
    for (unsigned idx = threadIdx.x; idx < args.input_volume; idx += blockDim.x) {
      offset = idx;
#pragma unroll
      for (int d = 0; d < DIM; d++) tile_point[d] = args.input_pitches[d].divmod(offset, offset);
      VAL value = in[input_bounds.lo + tile_point];
      // Write the value into shared memory
      input[idx] = value;
    }
  } else {
// Need to do point-wise tests
// Unroll this four times to try to pipeline loads
#pragma unroll 4
    for (unsigned idx = threadIdx.x; idx < args.input_volume; idx += blockDim.x) {
      offset = idx;
#pragma unroll
      for (int d = 0; d < DIM; d++) tile_point[d] = args.input_pitches[d].divmod(offset, offset);
      if (!root_rect.contains(input_bounds.lo + tile_point)) continue;
      VAL value = in[input_bounds.lo + tile_point];
      // Write the value into shared memory
      input[idx] = value;
    }
  }
  // Wait for everything to be loaded into shared memory
  __syncthreads();
  // Loop over points in the tile and compute the outputs
  coord_t f_coords[DIM];
  Point<DIM> out_point, in_point, filter_point;
  for (unsigned idx = threadIdx.x; idx < args.tile_volume; idx += blockDim.x) {
    // Compute the local coordinates
    offset = idx;
#pragma unroll
    for (int d = 0; d < DIM; d++) {
      tile_point[d] = args.block_pitches[d].divmod(offset, offset);
      out_point[d]  = block_point[d] + tile_point[d];
    }
    if (!subrect.contains(out_point)) continue;
#pragma unroll
    for (int d = 0; d < DIM; d++) f_coords[d] = 0;
    VAL acc{0};
    for (unsigned idx = 0; idx < args.filter_volume; idx++) {
#pragma unroll
      for (int d = 0; d < DIM; d++)
        in_point[d] = out_point[d] + f_coords[d] - args.filter_centers[d];
      if (input_contained || root_rect.contains(in_point)) {
        offset = 0;
#pragma unroll
        for (int d = 0; d < DIM; d++)
          offset += (tile_point[d] + f_coords[d]) * args.input_pitches[d].divisor;
#pragma unroll
        for (int d = 0; d < DIM; d++) filter_point[d] = args.filter_extents[d] - f_coords[d] - 1;
        acc = acc + input[offset] * filter[filter_point];
      }
// Step the filter coordinates
#pragma unroll
      for (int d = DIM - 1; d >= 0; d--) {
        f_coords[d]++;
        if (f_coords[d] == args.filter_extents[d])
          f_coords[d] = 0;
        else
          break;
      }
    }
    store_streaming(out.ptr(out_point), acc);
  }
}

// This version of the kernel is identical to the one above but with
// different launch bounds to handle a bigger CTA with more shared memory
template <typename VAL, int DIM>
__global__ static void __launch_bounds__(1024, 1)
  convolution_small_tile2(const AccessorWO<VAL, DIM> out,
                          const AccessorRO<VAL, DIM> filter,
                          const AccessorRO<VAL, DIM> in,
                          const Rect<DIM> root_rect,
                          const Rect<DIM> subrect,
                          const Rect<DIM> filter_rect,
                          const ConvolutionSmallTileArgs<DIM> args)
{
  // Deal with compiler shared memory stupidity
  extern __shared__ uint8_t buffer[];
  // Technically this illegal C++, but there's no other way to do it
  VAL* input = (VAL*)buffer;
  // Compute the origin point of the block
  size_t offset          = blockIdx.x;
  Point<DIM> block_point = subrect.lo;
#pragma unroll
  for (int d = 0; d < DIM; d++)
    block_point[d] += args.grid_pitches[d].divmod(offset, offset) * args.block_tiles[d];
  // Load in the shared memory for this block
  Point<DIM> tile_point;
  const Rect<DIM> input_bounds(block_point - args.delta_lo, block_point + args.delta_hi);
  const bool input_contained = root_rect.contains(input_bounds);
  if (input_contained) {
// All the points are contained, so no need for point-wise tests
// Unroll this four times to try to pipeline loads
#pragma unroll 4
    for (unsigned idx = threadIdx.x; idx < args.input_volume; idx += blockDim.x) {
      offset = idx;
#pragma unroll
      for (int d = 0; d < DIM; d++) tile_point[d] = args.input_pitches[d].divmod(offset, offset);
      VAL value = in[input_bounds.lo + tile_point];
      // Write the value into shared memory
      input[idx] = value;
    }
  } else {
// Need to do point-wise tests
// Unroll this four times to try to pipeline loads
#pragma unroll 4
    for (unsigned idx = threadIdx.x; idx < args.input_volume; idx += blockDim.x) {
      offset = idx;
#pragma unroll
      for (int d = 0; d < DIM; d++) tile_point[d] = args.input_pitches[d].divmod(offset, offset);
      if (!root_rect.contains(input_bounds.lo + tile_point)) continue;
      VAL value = in[input_bounds.lo + tile_point];
      // Write the value into shared memory
      input[idx] = value;
    }
  }
  // Wait for everything to be loaded into shared memory
  __syncthreads();
  // Loop over points in the tile and compute the outputs
  coord_t f_coords[DIM];
  Point<DIM> out_point, in_point, filter_point;
  for (unsigned idx = threadIdx.x; idx < args.tile_volume; idx += blockDim.x) {
    // Compute the local coordinates
    offset = idx;
#pragma unroll
    for (int d = 0; d < DIM; d++) {
      tile_point[d] = args.block_pitches[d].divmod(offset, offset);
      out_point[d]  = block_point[d] + tile_point[d];
    }
    if (!subrect.contains(out_point)) continue;
#pragma unroll
    for (int d = 0; d < DIM; d++) f_coords[d] = 0;
    VAL acc{0};
    for (unsigned idx = 0; idx < args.filter_volume; idx++) {
#pragma unroll
      for (int d = 0; d < DIM; d++)
        in_point[d] = out_point[d] + f_coords[d] - args.filter_centers[d];
      if (input_contained || root_rect.contains(in_point)) {
        offset = 0;
#pragma unroll
        for (int d = 0; d < DIM; d++)
          offset += (tile_point[d] + f_coords[d]) * args.input_pitches[d].divisor;
#pragma unroll
        for (int d = 0; d < DIM; d++) filter_point[d] = args.filter_extents[d] - f_coords[d] - 1;
        acc = acc + input[offset] * filter[filter_point];
      }
// Step the filter coordinates
#pragma unroll
      for (int d = DIM - 1; d >= 0; d--) {
        f_coords[d]++;
        if (f_coords[d] == args.filter_extents[d])
          f_coords[d] = 0;
        else
          break;
      }
    }
    store_streaming(out.ptr(out_point), acc);
  }
}

template <typename VAL, int DIM>
__host__ static inline void launch_small_tile_kernel(AccessorWO<VAL, DIM> out,
                                                     AccessorRO<VAL, DIM> filter,
                                                     AccessorRO<VAL, DIM> in,
                                                     const Rect<DIM>& root_rect,
                                                     const Rect<DIM>& subrect,
                                                     const Rect<DIM>& filter_rect,
                                                     const hipDeviceProp_t& properties,
                                                     const unsigned extents[DIM],
                                                     const unsigned centers[DIM],
                                                     Point<DIM>& tile,
                                                     unsigned smem_size,
                                                     size_t max_smem_size)
{
  // Make the tile as big as possible so that it fits in shared memory
  // Try to keep it rectangular to minimize surface-to-volume ratio
  // and improve the reuse of data
  // If the current tile is less than half the shared memory in the SM then
  // decrease the upper bound so we can get 2 CTAs/SM
  bool halved              = false;
  const unsigned half_smem = properties.sharedMemPerMultiprocessor / 2;
  if ((smem_size <= (half_smem)) && (half_smem < max_smem_size)) {
    max_smem_size = half_smem;
    halved        = true;
  }
  Point<DIM> padding;
  for (int d = 0; d < DIM; d++) padding[d] = 2 * centers[d];
  Point<DIM> bounds = subrect.hi - subrect.lo + Point<DIM>::ONES();
  smem_size         = roundup_tile<VAL, DIM>(tile, bounds, padding, max_smem_size);
  // At this point we've got the tile size that we're going to compute
  // and the amount of dynamic shared memory that we need
  // Compute the arguments needed for the kernel launch
  ConvolutionSmallTileArgs<DIM> args;
  size_t blocks        = 1;
  size_t tile_pitch    = 1;
  unsigned input_pitch = 1;
  args.filter_volume   = 1;
  for (int d = DIM - 1; d >= 0; d--) {
    size_t blocks_along_dim = ((subrect.hi[d] - subrect.lo[d]) + tile[d]) / tile[d];
    args.grid_pitches[d]    = FastDivmodU64(blocks);
    blocks *= blocks_along_dim;
    args.block_tiles[d]   = tile[d];
    args.block_pitches[d] = FastDivmodU64(tile_pitch);
    tile_pitch *= tile[d];
    args.delta_lo[d]      = centers[d];
    args.delta_hi[d]      = tile[d] + centers[d] - 1;
    args.input_pitches[d] = FastDivmodU64(input_pitch);
    input_pitch *= (args.delta_lo[d] + args.delta_hi[d] + 1);
    args.filter_centers[d] = centers[d];
    args.filter_extents[d] = extents[d];
    args.filter_volume *= extents[d];
  }
  args.tile_volume  = tile_pitch;
  args.input_volume = input_pitch;
  assert((input_pitch * sizeof(VAL)) == smem_size);
  if (halved) {
    if (tile_pitch < 512)
      convolution_small_tile1<VAL, DIM>
        <<<blocks, tile_pitch, smem_size>>>(out, filter, in, root_rect, subrect, filter_rect, args);
    else
      convolution_small_tile1<VAL, DIM>
        <<<blocks, 512, smem_size>>>(out, filter, in, root_rect, subrect, filter_rect, args);
  } else {
    if (tile_pitch < 1024)
      convolution_small_tile2<VAL, DIM>
        <<<blocks, tile_pitch, smem_size>>>(out, filter, in, root_rect, subrect, filter_rect, args);
    else
      convolution_small_tile2<VAL, DIM>
        <<<blocks, 1024, smem_size>>>(out, filter, in, root_rect, subrect, filter_rect, args);
  }
}

template <typename VAL, int32_t DIM>
__host__ void direct_convolution(AccessorWO<VAL, DIM> out,
                                 AccessorRO<VAL, DIM> filter,
                                 AccessorRO<VAL, DIM> in,
                                 const Rect<DIM>& root_rect,
                                 const Rect<DIM>& subrect,
                                 const Rect<DIM>& filter_rect)
{
  constexpr int THREADVALS = THREAD_OUTPUTS(VAL);
  // Get the maximum amount of shared memory per threadblock
  int device;
  CHECK_CUDA(hipGetDevice(&device));
  hipDeviceProp_t properties;
  CHECK_CUDA(hipGetDeviceProperties(&properties, device));
  size_t max_smem_size = properties.sharedMemPerBlockOptin;

  // Only need to do these calls the first time on each device so
  // we use a bit mask to track which devices we've done it for
  static unsigned long long mask = 0;
  if (!(mask & (1 << device))) {
    if (properties.sharedMemPerBlock < max_smem_size) {
      CHECK_CUDA(hipFuncSetAttribute(reinterpret_cast<const void*>(convolution_small_tile1<VAL), DIM>,
                                      hipFuncAttributeMaxDynamicSharedMemorySize,
                                      max_smem_size));
      CHECK_CUDA(hipFuncSetAttribute(reinterpret_cast<const void*>(convolution_small_tile2<VAL), DIM>,
                                      hipFuncAttributeMaxDynamicSharedMemorySize,
                                      max_smem_size));
      CHECK_CUDA(hipFuncSetAttribute(reinterpret_cast<const void*>(convolution_large_tile<VAL), DIM, THREADVALS>,
                                      hipFuncAttributeMaxDynamicSharedMemorySize,
                                      max_smem_size));
    }
    if (sizeof(VAL) >= 8) {
      // Only need to set this on the first invocation
      CHECK_CUDA(hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(convolution_small_tile1<VAL), DIM>,
                                            hipSharedMemBankSizeEightByte));
      CHECK_CUDA(hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(convolution_small_tile2<VAL), DIM>,
                                            hipSharedMemBankSizeEightByte));
      CHECK_CUDA(hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(convolution_large_tile<VAL), DIM, THREADVALS>,
                                            hipSharedMemBankSizeEightByte));
    }
    // Make sure we have enough bits for every device
    assert(device < (8 * sizeof(mask)));
    // Make sure not to race with updates from other GPUs
    __sync_fetch_and_add(&mask, (1 << device));
  }
  unsigned extents[DIM];
  unsigned centers[DIM];
  for (int d = 0; d < DIM; d++) {
    assert(filter_rect.lo[d] == 0);
    extents[d] = filter_rect.hi[d] + 1;
    centers[d] = static_cast<coord_t>(extents[d] / 2);
  }
  Point<DIM> tile;
  for (int d = DIM - 1; d >= 0; d--) {
    // Make sure that each tile is at least double the size of the filter
    // so that we can get some savings in bandwidth needed
    tile[d] = 2 * centers[d];
    if (d == (DIM - 1)) {
      // In order to maximize bandwidth, we want to make sure we're loading at
      // least 128B of contiguous memory along the last axis (row-major) of input
      const unsigned min_contig_elmts = 128 / sizeof(VAL);
      if ((tile[d] + 2 * centers[d]) < min_contig_elmts)
        tile[d] = min_contig_elmts - 2 * centers[d];
    }
  }
  unsigned smem_size = sizeof(VAL);
  for (int d = 0; d < DIM; d++) smem_size *= (tile[d] + 2 * centers[d]);
  if (smem_size <= max_smem_size) {
    // Small tile case:
    launch_small_tile_kernel<VAL, DIM>(out,
                                       filter,
                                       in,
                                       root_rect,
                                       subrect,
                                       filter_rect,
                                       properties,
                                       extents,
                                       centers,
                                       tile,
                                       smem_size,
                                       max_smem_size);
  } else {
    // Large tile case:
    // If we're going to do this, we need to initialize the output to zeros
    // so we can kick that off to the GPU while we figure out how to launch
    // the rest of the kernels to do the convolution
    size_t strides[DIM];
    VAL* out_ptr = out.ptr(subrect, strides);
    // Check to see if the output is dense
    bool out_dense   = true;
    size_t out_pitch = 1;
    for (int d = DIM - 1; d >= 0; d--) {
      if (strides[d] != out_pitch) {
        out_dense = false;
        break;
      }
      out_pitch *= strides[d];
    }
    if (out_dense) {
      size_t bytes = sizeof(VAL) * out_pitch;
      CHECK_CUDA(hipMemsetAsync(out_ptr, 0, bytes));
    } else {
      out_pitch = 1;
      ConvolutionInitArgs<DIM> args;
      for (int d = DIM - 1; d >= 0; d--) {
        args.pitches[d] = FastDivmodU64(out_pitch);
        out_pitch *= (subrect.hi[d] - subrect.lo[d] + 1);
      }
      size_t blocks = (out_pitch + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
      convolution_init<VAL, DIM><<<blocks, THREADS_PER_BLOCK>>>(out, subrect.lo, args, out_pitch);
    }
    // Figure out the shape of the L1 output tile based on the number of
    // points that we can fit into registers
    Point<DIM> l1_output_tile;
    const unsigned max_l1_output_volume = CONVOLUTION_THREADS * THREADVALS;
    // Make sure the max_l1_output_volume doesn't consume more than half of shared memory
    unsigned target_l1_output_volume = max_l1_output_volume;
    while ((max_smem_size / 2) < (target_l1_output_volume * sizeof(VAL)))
      target_l1_output_volume /= 2;
    const Point<DIM> output_bounds = subrect.hi - subrect.lo + Point<DIM>::ONES();
    const unsigned l1_output_volume =
      compute_output_tile<VAL, DIM>(l1_output_tile,
                                    output_bounds,
                                    128 /*cache line size*/ / sizeof(VAL),
                                    target_l1_output_volume);
    // At this point we've got our output tile, compute how big a filter
    // tile we can make and still fit both the filter tile and the
    // input tile into the maximum amount of shared memory for this GPU
    Point<DIM> l1_filter_tile;
    const Point<DIM> filter_bounds = filter_rect.hi - filter_rect.lo + Point<DIM>::ONES();
    unsigned dynamic_smem =
      compute_filter_tile<VAL, DIM>(l1_filter_tile, filter_bounds, l1_output_tile, max_smem_size);
    unsigned input_smem_offset = 1;
    for (int d = 0; d < DIM; d++) input_smem_offset *= l1_filter_tile[d];
    // Tile the number of SMs on this GPU to compute the shape of the
    // L2 output tile for this kernel
    // We assume here that the number of SMs is easily factorable
    // into primes of 2, 3, and 5. It would be strange if we have a
    // GPU with a number of SMs these days that can't be factored
    // this way. If we do report a warning.
    unsigned l2_tiles[DIM];
    for (int d = 0; d < DIM; d++) l2_tiles[d] = 1;
    if (DIM > 1) {
      unsigned twos = 0, threes = 0, fives = 0;
      unsigned remainder = properties.multiProcessorCount;
      while ((remainder > 1) && ((remainder % 2) == 0)) {
        twos++;
        remainder /= 2;
      }
      while ((remainder > 1) && ((remainder % 3) == 0)) {
        threes++;
        remainder /= 3;
      }
      while ((remainder > 1) && ((remainder % 5) == 0)) {
        fives++;
        remainder /= 5;
      }
      if (remainder > 1) {
        fprintf(stdout,
                "WARNING: %d is an unusual number of SMs "
                "for GPU convolution. Please report your GPU kind and "
                "the number of SMs in a Legate NumPy issue.",
                properties.multiProcessorCount);
        l2_tiles[DIM - 1] = remainder;
      }
      for (unsigned idx = 0; idx < fives; idx++) {
        int smallest = 0;
        for (int d = 1; d < DIM; d++) {
          if (l2_tiles[smallest] < l2_tiles[d]) continue;
          smallest = d;
        }
        l2_tiles[smallest] *= 5;
      }
      for (unsigned idx = 0; idx < threes; idx++) {
        int smallest = 0;
        for (int d = 1; d < DIM; d++) {
          if (l2_tiles[smallest] < l2_tiles[d]) continue;
          smallest = d;
        }
        l2_tiles[smallest] *= 3;
      }
      for (unsigned idx = 0; idx < twos; idx++) {
        int smallest = 0;
        for (int d = 1; d < DIM; d++) {
          if (l2_tiles[smallest] < l2_tiles[d]) continue;
          smallest = d;
        }
        l2_tiles[smallest] *= 2;
      }
    } else {
      l2_tiles[0] = properties.multiProcessorCount;
    }
    // Now that we've got a tiling of the l1 output blocks across
    // the SMs compute how big it is in memory and see if it is less
    // than a quarter of the L2 cache so we can block for the L2
    Point<DIM> l2_output_tile;
    size_t l2_output_tile_size = sizeof(VAL);
    for (int d = 0; d < DIM; d++) {
      l2_output_tile[d] = l2_tiles[d] * l1_output_tile[d];
      l2_output_tile_size *= l2_output_tile[d];
    }
    Point<DIM> l2_filter_tile;
    size_t total_l2_filters = 1;
    if (l2_output_tile_size <= (properties.l2CacheSize / 4)) {
      for (int d = 0; d < DIM; d++) l2_filter_tile[d] = 1;
      // Compute the L2 filter tile size so that the L2 filter and the
      // corresponding L2 input tile will fit in the L2 cache
      compute_filter_tile<VAL, DIM>(
        l2_filter_tile, filter_bounds, l2_output_tile, 3 * properties.l2CacheSize / 4);
      for (int d = 0; d < DIM; d++)
        total_l2_filters *= (filter_bounds[d] + l2_filter_tile[d] - 1) / l2_filter_tile[d];
    } else {
      // It's likely this tile is too big to block for the L2 cache
      // so we're not going to bother blocking for the L2 and just
      // run everything out of the framebuffer memory. The upside is
      // that we'll only need to make a single pass over the input
      for (int d = 0; d < DIM; d++) l2_filter_tile[d] = filter_rect.hi[d] - filter_rect.lo[d] + 1;
    }
    // Construct the arguments for the kernel launches
    ConvolutionLargeTileArgs<DIM, THREADVALS> args;
    int pitch = 1;
    for (int d = DIM - 1; d >= 0; d--) {
      args.l1_input_pitches[d] = FastDivmod(pitch);
      pitch *= (l1_output_tile[d] + 2 * (l1_filter_tile[d] / 2));
    }
    pitch = 1;
    for (int d = DIM - 1; d >= 0; d--) {
      args.l1_filter_pitches[d] = FastDivmod(pitch);
      pitch *= l1_filter_tile[d];
    }
    pitch = 1;
    for (int d = DIM - 1; d >= 0; d--) {
      args.l1_output_pitches[d] = FastDivmod(pitch);
      pitch *= l1_output_tile[d];
    }
    args.l2_output_tile      = l2_output_tile;
    args.l2_filter_tile      = l2_filter_tile;
    args.l1_output_tile      = l1_output_tile;
    args.l1_filter_tile      = l1_filter_tile;
    args.l2_output_limits    = output_bounds;
    args.shared_input_offset = input_smem_offset;
    args.total_l2_outputs    = 1;
    args.total_l1_outputs    = 1;
    args.total_l1_filters    = 1;
    args.l1_filter_points    = 1;
    args.l1_input_points     = 1;
    pitch                    = 1;
    for (int d = DIM - 1; d >= 0; d--) {
      args.total_l2_outputs *= (output_bounds[d] + l2_output_tile[d] - 1) / l2_output_tile[d];
      args.l1_output_tile_pitches[d] = FastDivmod(pitch);
      pitch *= (l2_output_tile[d] + l1_output_tile[d] - 1) / l1_output_tile[d];
      args.total_l1_filters *= (l2_filter_tile[d] + l1_filter_tile[d] - 1) / l1_filter_tile[d];
      args.l1_filter_points *= l1_filter_tile[d];
      args.l1_input_points *= (l1_output_tile[d] + 2 * (l1_filter_tile[d] / 2));
    }
    args.total_l1_outputs = pitch;
    // Figure out how to tile the points across the l1_output_tile
    if (DIM > 1) {
      unsigned regsteps[DIM];
      for (int d = 0; d < DIM; d++) regsteps[d] = 0;
      unsigned remainder = THREADVALS;
      // Handle the case here where we aren't going to use all
      // the points in the registers so we need to scale back
      if (l1_output_volume < max_l1_output_volume) {
        assert((max_l1_output_volume % l1_output_volume) == 0);
        remainder /= (max_l1_output_volume / l1_output_volume);
        if (remainder == 0) remainder = 1;
      }
      for (int d = 0; d < DIM; d++) {
        if (remainder == 1) {
          regsteps[d] = l1_output_tile[d];
        } else if (remainder <= l1_output_tile[d]) {
          // All powers of two so should always divide
          assert((l1_output_tile[d] % remainder) == 0);
          regsteps[d] = l1_output_tile[d] / remainder;
          remainder   = 1;
        } else {
          // All powers of two so should always divide
          assert((remainder % l1_output_tile[d]) == 0);
          regsteps[d] = 1;
          remainder /= l1_output_tile[d];
        }
      }
      assert(remainder == 1);
      Point<DIM, unsigned> offset = Point<DIM, unsigned>::ZEROES();
      for (int p = 0; p < THREADVALS; p++) {
        args.point_offsets[p] = offset;
        // Step to the next offset
        for (int d = DIM - 1; d >= 0; d--) {
          offset[d] += regsteps[d];
          if (offset[d] == l1_output_tile[d]) {
            if ((d == 0) && (p != (THREADVALS - 1)))
              // Allow overflow in this case to handle the case
              // where we have more points than we need for the l1 output tile
              assert(l1_output_volume < max_l1_output_volume);
            else
              offset[d] = 0;
          } else
            break;
        }
      }
      args.uniform_input_stride = regsteps[0] * args.l1_input_pitches[0].divisor;
      // Check to make sure this is the uniform input stride case
      for (int d = 1; d < DIM; d++) {
        if (regsteps[d] == l1_output_tile[d]) continue;
        args.uniform_input_stride = 0;
        break;
      }
    } else {
      assert(THREADVALS <= l1_output_tile[0]);
      unsigned remainder = THREADVALS;
      // Handle the case here where we aren't going to use all
      // the points in the registers so we need to scale back
      if (l1_output_volume < max_l1_output_volume) {
        assert((max_l1_output_volume % l1_output_volume) == 0);
        remainder /= (max_l1_output_volume / l1_output_volume);
        if (remainder == 0) remainder = 1;
      }
      assert((l1_output_tile[0] % remainder) == 0);
      unsigned regstep = l1_output_tile[0] / remainder;
      for (int p = 0; p < THREADVALS; p++) args.point_offsets[p][0] = p * regstep;
      args.uniform_input_stride = regstep * args.l1_input_pitches[0].divisor;
    }
    if (l1_output_volume < max_l1_output_volume) {
      args.shared_input_bound = dynamic_smem / sizeof(VAL);
      args.total_l1_points    = l1_output_volume;
    } else {
      args.shared_input_bound = 0;
      args.total_l1_points    = 0;
    }
    // Launch as many kernels as we need to walk over the entire filter
    // Given the L2 filter tile that we came up with
    const Point<DIM, unsigned> zero = Point<DIM, unsigned>::ZEROES();
    const Point<DIM, unsigned> one  = Point<DIM, unsigned>::ONES();
    if (total_l2_filters > 1) {
      Point<DIM> l2_filter_lo = filter_rect.lo;
      for (unsigned idx = 0; idx < total_l2_filters; idx++) {
        Rect<DIM> l2_filter_rect(l2_filter_lo, l2_filter_lo + l2_filter_tile - one);
        l2_filter_rect = l2_filter_rect.intersection(filter_rect);
        const Point<DIM> l1_input_start =
          subrect.lo + Point<DIM>(extents) - l2_filter_lo - l1_filter_tile - Point<DIM>(centers);
        const Point<DIM> l2_input_start =
          subrect.lo + Point<DIM>(extents) - l2_filter_rect.hi - one - Point<DIM>(centers);
        const Point<DIM> l2_input_stop = subrect.lo + l2_output_tile - one + Point<DIM>(extents) -
                                         l2_filter_rect.lo - one - Point<DIM>(centers);
        convolution_large_tile<VAL, DIM, THREADVALS>
          <<<properties.multiProcessorCount, CONVOLUTION_THREADS, dynamic_smem>>>(out,
                                                                                  filter,
                                                                                  in,
                                                                                  root_rect,
                                                                                  subrect,
                                                                                  l2_filter_rect,
                                                                                  l2_input_start,
                                                                                  l2_input_stop,
                                                                                  l1_input_start,
                                                                                  zero,
                                                                                  one,
                                                                                  args);
        // Step to the next filter
        for (int d = DIM - 1; d >= 0; d--) {
          l2_filter_lo[d] += l2_filter_tile[d];
          if (filter_rect.hi[d] < l2_filter_lo[d])
            l2_filter_lo[d] = filter_rect.lo[d];
          else
            break;
        }
      }
    } else {
      assert(total_l2_filters == 1);
      const Point<DIM> l1_input_start =
        subrect.lo + Point<DIM>(extents) - filter_rect.lo - l1_filter_tile - Point<DIM>(centers);
      const Point<DIM> l2_input_start = subrect.lo - Point<DIM>(centers);
      const Point<DIM> l2_input_stop  = subrect.lo + l2_output_tile - one + Point<DIM>(extents) -
                                       filter_rect.lo - one - Point<DIM>(centers);
      convolution_large_tile<VAL, DIM, THREADVALS>
        <<<properties.multiProcessorCount, CONVOLUTION_THREADS, dynamic_smem>>>(out,
                                                                                filter,
                                                                                in,
                                                                                root_rect,
                                                                                subrect,
                                                                                filter_rect,
                                                                                l2_input_start,
                                                                                l2_input_stop,
                                                                                l1_input_start,
                                                                                zero,
                                                                                one,
                                                                                args);
    }
  }
}

///////////////////////////////////////
// FFT-based convolution implementation
///////////////////////////////////////

template <typename T>
class Shadow {
 public:
  template <typename... Fnargs>
  bool update(Fnargs&&... args)
  {
    dirty_ = host_.update(std::forward<Fnargs>(args)...);
    return dirty_;
  }
  T& host() { return host_; }
  T* device(hipStream_t stream)
  {
    if (nullptr == device_) {
      CHECK_CUDA(hipMalloc(&device_, sizeof(T)));
      dirty_ = true;
    }
    if (dirty_) {
      CHECK_CUDA(hipMemcpyAsync(device_, &host_, sizeof(T), hipMemcpyHostToDevice, stream));
      dirty_ = false;
    }
    assert(device_ != nullptr);
    return device_;
  }

 private:
  bool dirty_{true};
  T host_{};
  T* device_{nullptr};
};

template <void* (*F)()>
class Cache {
 public:
  void* operator()()
  {
    if (nullptr == cache_) cache_ = F();
    return cache_;
  };

 private:
  void* cache_{nullptr};
};

template <typename VAL>
struct ForwardPass;

template <>
struct ForwardPass<float> {
  static constexpr hipfftType type = HIPFFT_R2C;
  static constexpr hipfftXtCallbackType callback_type(bool load)
  {
    return load ? HIPFFT_CB_LD_REAL : HIPFFT_CB_ST_COMPLEX;
  }
  static __host__ inline void execute(hipfftHandle plan, const float* idata, float* odata)
  {
    CHECK_CUFFT(hipfftExecR2C(plan, (hipfftReal*)idata, (hipfftComplex*)odata));
  }
};

template <>
struct ForwardPass<double> {
  static constexpr hipfftType type = HIPFFT_D2Z;
  static constexpr hipfftXtCallbackType callback_type(bool load)
  {
    return load ? HIPFFT_CB_LD_REAL_DOUBLE : HIPFFT_CB_ST_COMPLEX_DOUBLE;
  }
  static __host__ inline void execute(hipfftHandle plan, const double* idata, double* odata)
  {
    CHECK_CUFFT(hipfftExecD2Z(plan, (hipfftDoubleReal*)idata, (hipfftDoubleComplex*)odata));
  }
};

template <typename VAL>
struct BackwardPass;

template <>
struct BackwardPass<float> {
  static constexpr hipfftType type = HIPFFT_C2R;
  static constexpr hipfftXtCallbackType callback_type(bool load)
  {
    return load ? HIPFFT_CB_LD_COMPLEX : HIPFFT_CB_ST_REAL;
  }
  static __host__ inline void execute(hipfftHandle plan, float* idata, float* odata)
  {
    CHECK_CUFFT(hipfftExecC2R(plan, (hipfftComplex*)idata, (hipfftReal*)odata));
  }
};

template <>
struct BackwardPass<double> {
  static constexpr hipfftType type = HIPFFT_Z2D;
  static constexpr hipfftXtCallbackType callback_type(bool load)
  {
    return load ? HIPFFT_CB_LD_COMPLEX_DOUBLE : HIPFFT_CB_ST_REAL_DOUBLE;
  }
  static __host__ inline void execute(hipfftHandle plan, double* idata, double* odata)
  {
    CHECK_CUFFT(hipfftExecZ2D(plan, (hipfftDoubleComplex*)idata, (hipfftDoubleReal*)odata));
  }
};

extern __host__ void* load_zero_pad_callback_float();
extern __host__ void* load_zero_pad_callback_double();

extern __host__ void* load_multiply_callback_float();
extern __host__ void* load_multiply_callback_double();

extern __host__ void* load_store_callback_float();
extern __host__ void* load_store_callback_double();

template <typename VAL>
struct Callbacks;

template <>
struct Callbacks<float> {
  static __host__ inline void* zero_pad() { return load_zero_pad_callback_float(); }
  static __host__ inline void* multiply() { return load_multiply_callback_float(); }
  static __host__ inline void* store() { return load_store_callback_float(); }
};

template <>
struct Callbacks<double> {
  static __host__ inline void* zero_pad() { return load_zero_pad_callback_double(); }
  static __host__ inline void* multiply() { return load_multiply_callback_double(); }
  static __host__ inline void* store() { return load_store_callback_double(); }
};

template <typename VAL, int DIM>
__host__ static inline void cufft_convolution(AccessorWO<VAL, DIM> out,
                                              AccessorRO<VAL, DIM> filter,
                                              AccessorRO<VAL, DIM> in,
                                              const Rect<DIM>& root_rect,
                                              const Rect<DIM>& subrect,
                                              const Rect<DIM>& filter_rect)
{
  int device;
  CHECK_CUDA(hipGetDevice(&device));
  hipDeviceProp_t properties;
  CHECK_CUDA(hipGetDeviceProperties(&properties, device));
  size_t max_smem_size = properties.sharedMemPerBlockOptin;

  // Only need to do these calls the first time on each device so
  // we use a bit mask to track which devices we've done it for
  static unsigned long long mask = 0;
  if (!(mask & (1 << device))) {
    if (properties.sharedMemPerBlock < max_smem_size) {
      CHECK_CUDA(hipFuncSetAttribute(reinterpret_cast<const void*>(convolution_small_tile1<VAL), DIM>,
                                      hipFuncAttributeMaxDynamicSharedMemorySize,
                                      max_smem_size));
      CHECK_CUDA(hipFuncSetAttribute(reinterpret_cast<const void*>(convolution_small_tile2<VAL), DIM>,
                                      hipFuncAttributeMaxDynamicSharedMemorySize,
                                      max_smem_size));
    }
    if (sizeof(VAL) >= 8) {
      // Only need to set this on the first invocation
      CHECK_CUDA(hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(convolution_small_tile1<VAL), DIM>,
                                            hipSharedMemBankSizeEightByte));
      CHECK_CUDA(hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(convolution_small_tile2<VAL), DIM>,
                                            hipSharedMemBankSizeEightByte));
    }
    // Make sure we have enough bits for every device
    assert(device < (8 * sizeof(mask)));
    // Make sure not to race with updates from other GPUs
    __sync_fetch_and_add(&mask, (1 << device));
  }
  unsigned extents[DIM];
  unsigned centers[DIM];
  for (int d = 0; d < DIM; d++) {
    assert(filter_rect.lo[d] == 0);
    extents[d] = filter_rect.hi[d] + 1;
    centers[d] = static_cast<coord_t>(extents[d] / 2);
  }
  Point<DIM> tile;
  for (int d = DIM - 1; d >= 0; d--) {
    // Make sure that each tile is at least double the size of the filter
    // so that we can get some savings in bandwidth needed
    tile[d] = 2 * centers[d];
    if (d == (DIM - 1)) {
      // In order to maximize bandwidth, we want to make sure we're loading at
      // least 128B of contiguous memory along the last axis (row-major) of input
      const unsigned min_contig_elmts = 128 / sizeof(VAL);
      if ((tile[d] + 2 * centers[d]) < min_contig_elmts)
        tile[d] = min_contig_elmts - 2 * centers[d];
    }
  }
  unsigned smem_size = sizeof(VAL);
  for (int d = 0; d < DIM; d++) smem_size *= (tile[d] + 2 * centers[d]);
  if (smem_size <= max_smem_size) {
    launch_small_tile_kernel<VAL, DIM>(out,
                                       filter,
                                       in,
                                       root_rect,
                                       subrect,
                                       filter_rect,
                                       properties,
                                       extents,
                                       centers,
                                       tile,
                                       smem_size,
                                       max_smem_size);
  } else {
    // Cache for metadata allocations
    struct Metadata {
      Shadow<ZeroPadLoadData> filter_meta;
      Shadow<ZeroPadLoadData> signal_meta;
      Shadow<LoadComplexData> load_meta;
      Shadow<StoreOutputData<VAL>> store_meta;
      Cache<Callbacks<VAL>::zero_pad> zero_pad;
      Cache<Callbacks<VAL>::multiply> multiply;
      Cache<Callbacks<VAL>::store> store;
    };
    static Metadata metadata_cache[LEGION_MAX_NUM_PROCS];

    // Instead of doing the large tile case, we can instead do this
    // by transforming both the input and the filter to the frequency
    // domain using an FFT, perform the convolution with a point-wise
    // multiplication, and then transform the result back to the spatial domain

    auto stream  = get_cached_stream();
    auto proc_id = Processor::get_executing_processor().id & (LEGION_MAX_NUM_PROCS - 1);
    auto& cache  = metadata_cache[proc_id];

    // First compute how big our temporary allocation needs to be
    // We'll need two of them to store the zero-padded data for the inputs
    const Point<DIM> zero = Point<DIM>::ZEROES();
    const Point<DIM> one  = Point<DIM>::ONES();
    Rect<DIM> offset_bounds;
    for (int d = 0; d < DIM; d++) {
      offset_bounds.lo[d] = subrect.lo[d] - centers[d];
      offset_bounds.hi[d] = subrect.hi[d] + extents[d] - 1 - centers[d];
    }
    Rect<DIM> input_bounds         = root_rect.intersection(offset_bounds);
    const Point<DIM> signal_bounds = input_bounds.hi - input_bounds.lo + one;
    const Point<DIM> filter_bounds = filter_rect.hi - filter_rect.lo + one;
    Point<DIM> fftsize             = signal_bounds + filter_bounds;
    // Technically we can shrink this by one and still be sound but we'll
    // only do that if it will make the number even
    for (int d = 0; d < DIM; d++) fftsize[d] -= fftsize[d] % 2;
    // Cufft needs the last dimension to have fftsize/2+1 complex elements for
    // the temporary buffer, we know that the last dimension is already even
    // so we just need to add two elements to the last dim of the fftsize
    Point<DIM> buffersize = fftsize;
    buffersize[DIM - 1] += 2;
    size_t buffervolume = 1;
    for (int d = 0; d < DIM; d++) buffervolume *= buffersize[d];
    // In theory we could do this with a single output buffer by doing
    // += operations in the second forward FFT kernel into the buffer,
    // but unfortunately cufft likes to use the output buffer during its
    // execution and that destroys the data from the first FFT
    DeferredBuffer<VAL, 1> buffer(Rect<1>(Point<1>(0), Point<1>(2 * buffervolume - 1)),
                                  Memory::GPU_FB_MEM,
                                  nullptr /*initial*/,
                                  128 /*alignment*/);
    VAL* buffer_ptr = buffer.ptr(Point<1>(0));

    auto forward_plan  = get_cufft_plan(ForwardPass<VAL>::type, fftsize);
    auto backward_plan = get_cufft_plan(BackwardPass<VAL>::type, fftsize);

    // Set the stream and working area for the plans
    CHECK_CUFFT(hipfftSetStream(forward_plan.handle(), stream));
    CHECK_CUFFT(hipfftSetStream(backward_plan.handle(), stream));

    auto workarea_size = std::max(forward_plan.workarea_size(), backward_plan.workarea_size());

    // Create the plan and allocate a temporary buffer for it if it needs one
    DeferredBuffer<uint8_t, 1> workarea_buffer;
    if (workarea_size > 0) {
      const Point<1> zero1d(0);
      workarea_buffer = DeferredBuffer<uint8_t, 1>(Rect<1>(zero1d, Point<1>(workarea_size - 1)),
                                                   Memory::GPU_FB_MEM,
                                                   nullptr /*initial*/,
                                                   128 /*alignment*/);
      void* workarea  = workarea_buffer.ptr(zero1d);
      CHECK_CUFFT(hipfftSetWorkArea(forward_plan.handle(), workarea));
      CHECK_CUFFT(hipfftSetWorkArea(backward_plan.handle(), workarea));
    }

    // FFT the filter data
    size_t strides[DIM];
    const auto* filter_ptr = filter.ptr(filter_rect, strides);
    cache.filter_meta.update(fftsize, strides, filter_bounds);

    auto* d_filter_meta = cache.filter_meta.device(stream);
    forward_plan.set_callback(
      ForwardPass<VAL>::callback_type(true), cache.zero_pad(), d_filter_meta);
    ForwardPass<VAL>::execute(forward_plan.handle(), filter_ptr, buffer_ptr);

    // FFT the input data
    const auto* signal_ptr = in.ptr(input_bounds, strides);
    cache.signal_meta.update(fftsize, strides, signal_bounds);

    auto* d_signal_meta = cache.signal_meta.device(stream);
    forward_plan.set_callback(
      ForwardPass<VAL>::callback_type(true), cache.zero_pad(), d_signal_meta);
    ForwardPass<VAL>::execute(forward_plan.handle(), signal_ptr, buffer_ptr + buffervolume);

    // Inverse FFT for the output in-place in the temporary buffer
    auto* output_ptr = out.ptr(subrect, strides);

    Point<DIM> offsets;
    for (int32_t d = 0; d < DIM; d++)
      offsets[d] =
        centers[d] - (1 - (extents[d] % 2)) +
        ((offset_bounds.lo[d] < root_rect.lo[d]) ? (subrect.lo[d] - root_rect.lo[d]) : centers[d]);
    auto output_bounds = subrect.hi - subrect.lo + one;

    cache.load_meta.update(buffervolume / 2);
    cache.store_meta.update(fftsize, strides, offsets, output_bounds);

    auto* d_load_meta  = cache.load_meta.device(stream);
    auto* d_store_meta = cache.store_meta.device(stream);
    backward_plan.set_callback(
      BackwardPass<VAL>::callback_type(true), cache.multiply(), d_load_meta);
    backward_plan.set_callback(
      BackwardPass<VAL>::callback_type(false), cache.store(), d_store_meta);
    BackwardPass<VAL>::execute(backward_plan.handle(), buffer_ptr, output_ptr);

    // This is useful debugging code for finding the output
#if 0
    {
      std::vector<VAL> vec_debug_buffer(buffervolume);
      auto* debug_buffer = vec_debug_buffer.data();
      CHECK_CUDA(hipMemcpyAsync(debug_buffer,
                                 buffer_ptr + buffervolume,
                                 buffervolume * sizeof(VAL),
                                 hipMemcpyDeviceToHost,
                                 stream));
      CHECK_CUDA(hipStreamSynchronize(stream));
      for (size_t idx = 0; idx < buffervolume; idx++) {
        if ((idx % fftsize[DIM - 1]) == 0) printf("\n");
        printf("%.8g ", debug_buffer[idx] * cache.store_meta.host().scale_factor);
      }
      printf("\n");
    }
#endif
  }
}

/////////////
// Dispatcher
/////////////

template <typename VAL, int DIM>
struct UseCUFFT {
  static constexpr bool value = 1 <= DIM && DIM <= 3 && std::is_floating_point<VAL>::value;
};

template <LegateTypeCode CODE, int DIM>
struct ConvolveImplBody<VariantKind::GPU, CODE, DIM> {
  using VAL = legate_type_of<CODE>;

  template <typename _VAL, int32_t _DIM, std::enable_if_t<UseCUFFT<_VAL, _DIM>::value>* = nullptr>
  __host__ void dispatch(AccessorWO<_VAL, _DIM> out,
                         AccessorRO<_VAL, _DIM> filter,
                         AccessorRO<_VAL, _DIM> in,
                         const Rect<_DIM>& root_rect,
                         const Rect<_DIM>& subrect,
                         const Rect<_DIM>& filter_rect) const
  {
    cufft_convolution<_VAL, _DIM>(out, filter, in, root_rect, subrect, filter_rect);
  }

  template <typename _VAL, int32_t _DIM, std::enable_if_t<!UseCUFFT<_VAL, _DIM>::value>* = nullptr>
  __host__ void dispatch(AccessorWO<_VAL, _DIM> out,
                         AccessorRO<_VAL, _DIM> filter,
                         AccessorRO<_VAL, _DIM> in,
                         const Rect<_DIM>& root_rect,
                         const Rect<_DIM>& subrect,
                         const Rect<_DIM>& filter_rect) const
  {
    direct_convolution<_VAL, _DIM>(out, filter, in, root_rect, subrect, filter_rect);
  }

  __host__ void operator()(AccessorWO<VAL, DIM> out,
                           AccessorRO<VAL, DIM> filter,
                           AccessorRO<VAL, DIM> in,
                           const Rect<DIM>& root_rect,
                           const Rect<DIM>& subrect,
                           const Rect<DIM>& filter_rect) const
  {
    dispatch(out, filter, in, root_rect, subrect, filter_rect);
  }
};

/*static*/ void ConvolveTask::gpu_variant(TaskContext& context)
{
  convolve_template<VariantKind::GPU>(context);
}

}  // namespace cunumeric
