#include "hip/hip_runtime.h"
/* Copyright 2021-2022 NVIDIA Corporation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */

#include "cunumeric/divmod.h"
#include "cunumeric/cuda_help.h"
#include "cunumeric/convolution/convolve.h"
#include "cunumeric/convolution/convolve_template.inl"

namespace cunumeric {

using namespace legate;

////////////////////////////////////
// Direct convolution implementation
////////////////////////////////////

// Convolution should be able to hit FMA throughput limits
// on the GPU due to the amount of FLOPs needed to be performed
// given the amount of data loaded. This is especially true of
// larger convolution filters. In order to hit these limits though
// we need to make sure that the GPU is fed data appropriately.
// We have two different kernels to handle different sized filters.

// Small Tile Case
// In the small tile case, a reasonable tile input including the
// all the boundary values for a given filter tile can fit in the
// shared memory of the SM, allowing the threadblock to fully
// compute an entire tile of output points in a single pass.
// If the tile is small enough, we even try to get multiple CTAs/SM
// in order to better pipeline data loading with compute.

// Large Tile Case
// For inputs where the filter is very large and it is impossible
// to fit a reasonable sized tile into shared memory, we tile both
// the output and the filter and make multiple passes over the data
// to create reasonable sized input tiles that fit in shared memory.
// If possible we also attempt to tile for the L2 cache as well so
// that threadblocks walking through memory together can hopefully
// hit in the L2 more often than not when loading data

template <int DIM>
struct ConvolutionInitArgs {
 public:
  FastDivmodU64 pitches[DIM];
};

template <typename VAL, int DIM>
__global__ static void __launch_bounds__(THREADS_PER_BLOCK, 4)
  convolution_init(const AccessorWO<VAL, DIM> out,
                   const Point<DIM> subrect_lo,
                   const ConvolutionInitArgs<DIM> args,
                   const size_t volume)
{
  size_t offset = blockIdx.x * blockDim.x + threadIdx.x;
  if (offset >= volume) return;
  Point<DIM> point = subrect_lo;
#pragma unroll
  for (int d = 0; d < DIM; d++) point[d] += args.pitches[d].divmod(offset, offset);
  out[point] = VAL{0};
}

// We want to run the convolution kernel with as large a shared memory
// tile as possible to avoid duplicate loading of data and maximize
// compute intensity. Therefore we're always going to run with 1 CTA
// per SM, but we still want enough thread-level parallelism, so we
// set this to the maximum number of warps in a threadblock
// Note that a lot of this code assumes this is a power of 2
#define CONVOLUTION_THREADS 1024
// The idea behind THREAD_OUTPUTS is to figure out how many registers
// we will be willing to assign to hold the partial output accumulations
// in each thread without using too many registers. Every GPU (with one
// exception) has 64K 32-bit registers per SM. We key off that and want
// to allocate a quarter of those registers for holding the partial accumulations
// We assume here that sizeof(VAL) is a power of 2
#define THREAD_OUTPUTS(TYPE) 1
//((4/*bytes/reg*/ * ((65536/8)/CONVOLUTION_THREADS)/*regs/thread*/) / sizeof(TYPE))

template <int DIM, int POINTS>
struct ConvolutionLargeTileArgs {
  FastDivmod l1_output_tile_pitches[DIM];
  FastDivmod l1_input_pitches[DIM];
  FastDivmod l1_filter_pitches[DIM];
  FastDivmod l1_output_pitches[DIM];
  Point<DIM> l2_output_limits;
  Point<DIM, unsigned> point_offsets[POINTS];
  Point<DIM, unsigned> l2_output_tile;
  Point<DIM, unsigned> l2_filter_tile;
  Point<DIM, unsigned> l1_output_tile;
  Point<DIM, unsigned> l1_filter_tile;
  unsigned total_l2_outputs;
  unsigned total_l1_outputs;
  unsigned total_l1_filters;
  unsigned total_l1_points;
  unsigned l1_filter_points;
  unsigned l1_input_points;
  unsigned shared_input_offset;
  unsigned uniform_input_stride;
  unsigned shared_input_bound;
};

template <typename VAL, int DIM, int POINTS>
__global__ static void __launch_bounds__(CONVOLUTION_THREADS, 1)
  convolution_large_tile(const AccessorWO<VAL, DIM> out,
                         const AccessorRO<VAL, DIM> filter,
                         const AccessorRO<VAL, DIM> in,
                         const Rect<DIM> root_rect,
                         const Rect<DIM> subrect,
                         const Rect<DIM> l2_filter_rect,
                         const Point<DIM> l2_input_start,
                         const Point<DIM> l2_input_stop,
                         const Point<DIM> l1_input_start,
                         const Point<DIM, unsigned> zero,
                         const Point<DIM, unsigned> one,
                         const ConvolutionLargeTileArgs<DIM, POINTS> args)
{
  // Deal with compiler shared memory stupidity
  extern __shared__ uint8_t buffer[];
  // Technically this is illegal C++, but there's no other way to do it
  VAL* sharedmem = (VAL*)buffer;
  Point<DIM, unsigned> thread_offset;
  int offset = threadIdx.x;
#pragma unroll
  for (int d = 0; d < DIM; d++) thread_offset[d] = args.l1_output_pitches[d].divmod(offset, offset);
  Point<DIM> l2_output_offset = zero;
  for (unsigned l2_outidx = 0; l2_outidx < args.total_l2_outputs; l2_outidx++) {
    // Do a quick check here to see if all the inputs are contained for this tile
    // l2_input_start = subrect.lo + args.extents - l2_filter_rect.hi - one - centers
    // l2_input_stop = subrect.lo + l2_output_tile - one + args.extents - l2_filter_rect.lo - one -
    // centers
    const Rect<DIM> l2_input_rect(l2_input_start + l2_output_offset,
                                  l2_input_stop + l2_output_offset);
    const bool input_contained = root_rect.contains(l2_input_rect);
    // Iterate the L1 output tiles that this threadblock should compute for the L2 output
    for (unsigned l1_outidx = blockIdx.x; l1_outidx < args.total_l1_outputs;
         l1_outidx += gridDim.x) {
      Point<DIM, unsigned> l1_output_offset;
      offset = l1_outidx;
#pragma unroll
      for (int d = 0; d < DIM; d++)
        l1_output_offset[d] =
          args.l1_output_tile_pitches[d].divmod(offset, offset) * args.l1_output_tile[d];
      // Handle the boundary case where an L1 tile is not contained in the L2 tile
      // becasue the L2 tile is overlapping a boundary. Note this decisions is the
      // same for all the threads in the threadblock so no bad divergence
      bool output_contained = true;
#pragma unroll
      for (int d = 0; d < DIM; d++) {
        if ((subrect.lo[d] + l2_output_offset[d] + l1_output_offset[d]) <= subrect.hi[d]) continue;
        output_contained = false;
        break;
      }
      if (!output_contained) continue;
      // Initialize our point data
      VAL acc[POINTS];
#pragma unroll
      for (int p = 0; p < POINTS; p++) acc[p] = VAL{0};
      // Iterate over the l1 filter tiles
      Point<DIM, unsigned> l1_filter_offset = zero;
      for (unsigned l1_fidx = 0; l1_fidx < args.total_l1_filters; l1_fidx++) {
        // Wait for any previous readers to be done
        __syncthreads();
// Load the filter into shared memory
// Unroll this a few times to get some memory level parallelims
#pragma unroll 4
        for (unsigned fidx = threadIdx.x; fidx < args.l1_filter_points; fidx += blockDim.x) {
          Point<DIM> filter_point = l2_filter_rect.lo + l1_filter_offset;
          offset                  = fidx;
#pragma unroll
          for (int d = 0; d < DIM; d++)
            filter_point[d] += args.l1_filter_pitches[d].divmod(offset, offset);
          if (l2_filter_rect.contains(filter_point))
            sharedmem[fidx] = filter[filter_point];
          else
            sharedmem[fidx] = VAL{0};
        }
        // Load the input into shared memory
        // Compute the input start point
        // input_start = subrect.lo + extents - l2_filter_rect.lo - l1_filter_tile - centers
        Point<DIM> input_start = l1_input_start + l2_output_offset + l1_output_offset;
        input_start -= l1_filter_offset;
// Unroll this a few times to get some memory level parallelism
#pragma unroll 4
        for (unsigned idx = threadIdx.x; idx < args.l1_input_points; idx += blockDim.x) {
          Point<DIM> input_point = input_start;
          offset                 = idx;
#pragma unroll
          for (int d = 0; d < DIM; d++)
            input_point[d] += args.l1_input_pitches[d].divmod(offset, offset);
          if (input_contained || root_rect.contains(input_point))
            sharedmem[args.shared_input_offset + idx] = in[input_point];
          else
            sharedmem[args.shared_input_offset + idx] = VAL{0};
        }
        // Wait for everything to be loaded into shared memory
        __syncthreads();
        // Iterate the points in the filter
        // We can safely iterate all the filter points and input points
        // because we wrote zeros into shared memory for everything that
        // was out of bounds
        Point<DIM, unsigned> filter_point = zero;
        if (args.uniform_input_stride) {
          // Each point is a constant offset in shared from the others
          unsigned input_offset = args.shared_input_offset;
#pragma unroll
          for (int d = 0; d < DIM; d++)
            input_offset +=
              args.l1_input_pitches[d].divisor * (thread_offset[d] + args.l1_filter_tile[d] - 1);
          if (args.shared_input_bound) {
            for (unsigned fidx = 0; fidx < args.l1_filter_points; fidx++) {
              // Use shared memory broadcasting functionality to avoid bank conflicts
              const VAL filter_value = sharedmem[fidx];
              unsigned point_offset  = input_offset;
#pragma unroll
              for (int p = 0; p < POINTS; p++) {
                if (args.shared_input_bound <= point_offset) break;
                acc[p] = acc[p] + filter_value * sharedmem[point_offset];
                point_offset += args.uniform_input_stride;
              }
// Step to the next filter point and update the input stride
#pragma unroll
              for (int d = DIM - 1; d >= 0; d--) {
                filter_point[d]++;
                input_offset -= args.l1_input_pitches[d].divisor;
                if (filter_point[d] == args.l1_filter_tile[d]) {
                  input_offset += args.l1_filter_tile[d] * args.l1_input_pitches[d].divisor;
                  filter_point[d] = 0;
                } else {
                  break;
                }
              }
            }
          } else {
            for (unsigned fidx = 0; fidx < args.l1_filter_points; fidx++) {
              // Use shared memory broadcasting functionality to avoid bank conflicts
              const VAL filter_value = sharedmem[fidx];
              unsigned point_offset  = input_offset;
#pragma unroll
              for (int p = 0; p < POINTS; p++) {
                acc[p] = acc[p] + filter_value * sharedmem[point_offset];
                point_offset += args.uniform_input_stride;
              }
// Step to the next filter point and update the input stride
#pragma unroll
              for (int d = DIM - 1; d >= 0; d--) {
                filter_point[d]++;
                input_offset -= args.l1_input_pitches[d].divisor;
                if (filter_point[d] == args.l1_filter_tile[d]) {
                  input_offset += args.l1_filter_tile[d] * args.l1_input_pitches[d].divisor;
                  filter_point[d] = 0;
                } else {
                  break;
                }
              }
            }
          }
        } else {
          // Need to compute the input offset uniquely for each point
          Point<DIM, unsigned> input_point = thread_offset + args.l1_filter_tile - one;
          unsigned point_offsets[POINTS];
#pragma unroll
          for (int p = 0; p < POINTS; p++) {
            point_offsets[p] = args.shared_input_offset;
#pragma unroll
            for (int d = 0; d < DIM; d++)
              point_offsets[p] +=
                (input_point[d] + args.point_offsets[p][d]) * args.l1_input_pitches[d].divisor;
          }
          unsigned filter_offset = 0;
          if (args.shared_input_bound) {
            for (unsigned fidx = 0; fidx < args.l1_filter_points; fidx++) {
              // Use shared memory broadcasting functionality to avoid bank conflicts
              const VAL filter_value = sharedmem[fidx];
#pragma unroll
              for (int p = 0; p < POINTS; p++) {
                unsigned point_offset = point_offsets[p] - filter_offset;
                if (args.shared_input_bound <= point_offset) continue;
                acc[p] = acc[p] + filter_value * sharedmem[point_offset];
              }
// Step to the next filter point
#pragma unroll
              for (int d = DIM - 1; d >= 0; d--) {
                filter_point[d]++;
                filter_offset += args.l1_input_pitches[d].divisor;
                if (filter_point[d] == args.l1_filter_tile[d]) {
                  filter_offset -= args.l1_filter_tile[d] * args.l1_input_pitches[d].divisor;
                  filter_point[d] = 0;
                } else {
                  break;
                }
              }
            }
          } else {
            for (unsigned fidx = 0; fidx < args.l1_filter_points; fidx++) {
              // Use shared memory broadcasting functionality to avoid bank conflicts
              const VAL filter_value = sharedmem[fidx];
#pragma unroll
              for (int p = 0; p < POINTS; p++) {
                unsigned point_offset = point_offsets[p] - filter_offset;
                acc[p]                = acc[p] + filter_value * sharedmem[point_offset];
              }
// Step to the next filter point
#pragma unroll
              for (int d = DIM - 1; d >= 0; d--) {
                filter_point[d]++;
                filter_offset += args.l1_input_pitches[d].divisor;
                if (filter_point[d] == args.l1_filter_tile[d]) {
                  filter_offset -= args.l1_filter_tile[d] * args.l1_input_pitches[d].divisor;
                  filter_point[d] = 0;
                } else {
                  break;
                }
              }
            }
          }
        }
// Step to the next L1 filter tile
#pragma unroll
        for (int d = DIM - 1; d >= 0; d--) {
          l1_filter_offset[d] += args.l1_filter_tile[d];
          if (args.l2_filter_tile[d] <= l1_filter_offset[d])
            l1_filter_offset[d] = 0;
          else
            break;
        }
      }
      // Now we can stream our accumulators back to the output
      Point<DIM> output = subrect.lo + l2_output_offset + (l1_output_offset + thread_offset);
      if (input_contained) {
        // If the input was contained, then so is the output
        if (args.total_l1_points) {
          unsigned index = threadIdx.x;
#pragma unroll
          for (int p = 0; p < POINTS; p++) {
            if (args.total_l1_points <= index) break;
            VAL* ptr = out.ptr(output + args.point_offsets[p]);
            // Make sure we don't pollute the L2 cache
            VAL value = load_streaming<VAL>(ptr);
            store_streaming<VAL>(ptr, value + acc[p]);
            index += blockDim.x;
          }
        } else {
#pragma unroll
          for (int p = 0; p < POINTS; p++) {
            VAL* ptr = out.ptr(output + args.point_offsets[p]);
            // Make sure we don't pollute the L2 cache
            VAL value = load_streaming<VAL>(ptr);
            store_streaming<VAL>(ptr, value + acc[p]);
          }
        }
      } else {
        // Input was not contained, so the output might not be either, do checks
        if (args.total_l1_points) {
          unsigned index = threadIdx.x;
#pragma unroll
          for (int p = 0; p < POINTS; p++) {
            if (args.total_l1_points <= index) break;
            Point<DIM> point = output + args.point_offsets[p];
            if (!subrect.contains(point)) break;
            VAL* ptr = out.ptr(point);
            // Make sure we don't pollute the L2 cache
            VAL value = load_streaming<VAL>(ptr);
            store_streaming<VAL>(ptr, value + acc[p]);
            index += blockDim.x;
          }
        } else {
#pragma unroll
          for (int p = 0; p < POINTS; p++) {
            Point<DIM> point = output + args.point_offsets[p];
            if (!subrect.contains(point)) continue;
            VAL* ptr = out.ptr(point);
            // Make sure we don't pollute the L2 cache
            VAL value = load_streaming<VAL>(ptr);
            store_streaming<VAL>(ptr, value + acc[p]);
          }
        }
      }
    }
// Step to the next output tile
#pragma unroll
    for (int d = DIM - 1; d >= 0; d--) {
      l2_output_offset[d] += args.l2_output_tile[d];
      if (args.l2_output_limits[d] <= l2_output_offset[d])
        l2_output_offset[d] = 0;
      else
        break;
    }
  }
}

template <int DIM>
struct ConvolutionSmallTileArgs {
  FastDivmodU64 grid_pitches[DIM];
  FastDivmodU64 block_pitches[DIM];
  FastDivmodU64 input_pitches[DIM];
  unsigned block_tiles[DIM];
  unsigned filter_centers[DIM];
  unsigned filter_extents[DIM];
  Point<DIM> delta_lo, delta_hi;
  size_t filter_volume;
  size_t tile_volume;
  size_t input_volume;
};

template <typename VAL, int DIM>
__global__ static void __launch_bounds__(512, 2)
  convolution_small_tile1(const AccessorWO<VAL, DIM> out,
                          const AccessorRO<VAL, DIM> filter,
                          const AccessorRO<VAL, DIM> in,
                          const Rect<DIM> root_rect,
                          const Rect<DIM> subrect,
                          const Rect<DIM> filter_rect,
                          const ConvolutionSmallTileArgs<DIM> args)
{
  // Deal with compiler shared memory stupidity
  extern __shared__ uint8_t buffer[];
  // Technically this illegal C++, but there's no other way to do it
  VAL* input = (VAL*)buffer;
  // Compute the origin point of the block
  size_t offset          = blockIdx.x;
  Point<DIM> block_point = subrect.lo;
#pragma unroll
  for (int d = 0; d < DIM; d++)
    block_point[d] += args.grid_pitches[d].divmod(offset, offset) * args.block_tiles[d];
  // Load in the shared memory for this block
  Point<DIM> tile_point;
  const Rect<DIM> input_bounds(block_point - args.delta_lo, block_point + args.delta_hi);
  const bool input_contained = root_rect.contains(input_bounds);
  if (input_contained) {
// All the points are contained, so no need for point-wise tests
// Unroll this four times to try to pipeline loads
#pragma unroll 4
    for (unsigned idx = threadIdx.x; idx < args.input_volume; idx += blockDim.x) {
      offset = idx;
#pragma unroll
      for (int d = 0; d < DIM; d++) tile_point[d] = args.input_pitches[d].divmod(offset, offset);
      VAL value = in[input_bounds.lo + tile_point];
      // Write the value into shared memory
      input[idx] = value;
    }
  } else {
// Need to do point-wise tests
// Unroll this four times to try to pipeline loads
#pragma unroll 4
    for (unsigned idx = threadIdx.x; idx < args.input_volume; idx += blockDim.x) {
      offset = idx;
#pragma unroll
      for (int d = 0; d < DIM; d++) tile_point[d] = args.input_pitches[d].divmod(offset, offset);
      if (!root_rect.contains(input_bounds.lo + tile_point)) continue;
      VAL value = in[input_bounds.lo + tile_point];
      // Write the value into shared memory
      input[idx] = value;
    }
  }
  // Wait for everything to be loaded into shared memory
  __syncthreads();
  // Loop over points in the tile and compute the outputs
  coord_t f_coords[DIM];
  Point<DIM> out_point, in_point, filter_point;
  for (unsigned idx = threadIdx.x; idx < args.tile_volume; idx += blockDim.x) {
    // Compute the local coordinates
    offset = idx;
#pragma unroll
    for (int d = 0; d < DIM; d++) {
      tile_point[d] = args.block_pitches[d].divmod(offset, offset);
      out_point[d]  = block_point[d] + tile_point[d];
    }
    if (!subrect.contains(out_point)) continue;
#pragma unroll
    for (int d = 0; d < DIM; d++) f_coords[d] = 0;
    VAL acc{0};
    for (unsigned idx = 0; idx < args.filter_volume; idx++) {
#pragma unroll
      for (int d = 0; d < DIM; d++)
        in_point[d] = out_point[d] + f_coords[d] - args.filter_centers[d];
      if (input_contained || root_rect.contains(in_point)) {
        offset = 0;
#pragma unroll
        for (int d = 0; d < DIM; d++)
          offset += (tile_point[d] + f_coords[d]) * args.input_pitches[d].divisor;
#pragma unroll
        for (int d = 0; d < DIM; d++) filter_point[d] = args.filter_extents[d] - f_coords[d] - 1;
        acc = acc + input[offset] * filter[filter_point];
      }
// Step the filter coordinates
#pragma unroll
      for (int d = DIM - 1; d >= 0; d--) {
        f_coords[d]++;
        if (f_coords[d] == args.filter_extents[d])
          f_coords[d] = 0;
        else
          break;
      }
    }
    store_streaming(out.ptr(out_point), acc);
  }
}

// This version of the kernel is identical to the one above but with
// different launch bounds to handle a bigger CTA with more shared memory
template <typename VAL, int DIM>
__global__ static void __launch_bounds__(1024, 1)
  convolution_small_tile2(const AccessorWO<VAL, DIM> out,
                          const AccessorRO<VAL, DIM> filter,
                          const AccessorRO<VAL, DIM> in,
                          const Rect<DIM> root_rect,
                          const Rect<DIM> subrect,
                          const Rect<DIM> filter_rect,
                          const ConvolutionSmallTileArgs<DIM> args)
{
  // Deal with compiler shared memory stupidity
  extern __shared__ uint8_t buffer[];
  // Technically this illegal C++, but there's no other way to do it
  VAL* input = (VAL*)buffer;
  // Compute the origin point of the block
  size_t offset          = blockIdx.x;
  Point<DIM> block_point = subrect.lo;
#pragma unroll
  for (int d = 0; d < DIM; d++)
    block_point[d] += args.grid_pitches[d].divmod(offset, offset) * args.block_tiles[d];
  // Load in the shared memory for this block
  Point<DIM> tile_point;
  const Rect<DIM> input_bounds(block_point - args.delta_lo, block_point + args.delta_hi);
  const bool input_contained = root_rect.contains(input_bounds);
  if (input_contained) {
// All the points are contained, so no need for point-wise tests
// Unroll this four times to try to pipeline loads
#pragma unroll 4
    for (unsigned idx = threadIdx.x; idx < args.input_volume; idx += blockDim.x) {
      offset = idx;
#pragma unroll
      for (int d = 0; d < DIM; d++) tile_point[d] = args.input_pitches[d].divmod(offset, offset);
      VAL value = in[input_bounds.lo + tile_point];
      // Write the value into shared memory
      input[idx] = value;
    }
  } else {
// Need to do point-wise tests
// Unroll this four times to try to pipeline loads
#pragma unroll 4
    for (unsigned idx = threadIdx.x; idx < args.input_volume; idx += blockDim.x) {
      offset = idx;
#pragma unroll
      for (int d = 0; d < DIM; d++) tile_point[d] = args.input_pitches[d].divmod(offset, offset);
      if (!root_rect.contains(input_bounds.lo + tile_point)) continue;
      VAL value = in[input_bounds.lo + tile_point];
      // Write the value into shared memory
      input[idx] = value;
    }
  }
  // Wait for everything to be loaded into shared memory
  __syncthreads();
  // Loop over points in the tile and compute the outputs
  coord_t f_coords[DIM];
  Point<DIM> out_point, in_point, filter_point;
  for (unsigned idx = threadIdx.x; idx < args.tile_volume; idx += blockDim.x) {
    // Compute the local coordinates
    offset = idx;
#pragma unroll
    for (int d = 0; d < DIM; d++) {
      tile_point[d] = args.block_pitches[d].divmod(offset, offset);
      out_point[d]  = block_point[d] + tile_point[d];
    }
    if (!subrect.contains(out_point)) continue;
#pragma unroll
    for (int d = 0; d < DIM; d++) f_coords[d] = 0;
    VAL acc{0};
    for (unsigned idx = 0; idx < args.filter_volume; idx++) {
#pragma unroll
      for (int d = 0; d < DIM; d++)
        in_point[d] = out_point[d] + f_coords[d] - args.filter_centers[d];
      if (input_contained || root_rect.contains(in_point)) {
        offset = 0;
#pragma unroll
        for (int d = 0; d < DIM; d++)
          offset += (tile_point[d] + f_coords[d]) * args.input_pitches[d].divisor;
#pragma unroll
        for (int d = 0; d < DIM; d++) filter_point[d] = args.filter_extents[d] - f_coords[d] - 1;
        acc = acc + input[offset] * filter[filter_point];
      }
// Step the filter coordinates
#pragma unroll
      for (int d = DIM - 1; d >= 0; d--) {
        f_coords[d]++;
        if (f_coords[d] == args.filter_extents[d])
          f_coords[d] = 0;
        else
          break;
      }
    }
    store_streaming(out.ptr(out_point), acc);
  }
}

template <typename VAL, int DIM>
__host__ static inline void launch_small_tile_kernel(AccessorWO<VAL, DIM> out,
                                                     AccessorRO<VAL, DIM> filter,
                                                     AccessorRO<VAL, DIM> in,
                                                     const Rect<DIM>& root_rect,
                                                     const Rect<DIM>& subrect,
                                                     const Rect<DIM>& filter_rect,
                                                     const hipDeviceProp_t& properties,
                                                     const unsigned extents[DIM],
                                                     const unsigned centers[DIM],
                                                     Point<DIM>& tile,
                                                     unsigned smem_size,
                                                     size_t max_smem_size)
{
  // Make the tile as big as possible so that it fits in shared memory
  // Try to keep it rectangular to minimize surface-to-volume ratio
  // and improve the reuse of data
  // If the current tile is less than half the shared memory in the SM then
  // decrease the upper bound so we can get 2 CTAs/SM
  bool halved              = false;
  const unsigned half_smem = properties.sharedMemPerMultiprocessor / 2;
  if ((smem_size <= (half_smem)) && (half_smem < max_smem_size)) {
    max_smem_size = half_smem;
    halved        = true;
  }
  Point<DIM> padding;
  for (int d = 0; d < DIM; d++) padding[d] = 2 * centers[d];
  Point<DIM> bounds = subrect.hi - subrect.lo + Point<DIM>::ONES();
  smem_size         = roundup_tile<VAL, DIM>(tile, bounds, padding, max_smem_size);
  // At this point we've got the tile size that we're going to compute
  // and the amount of dynamic shared memory that we need
  // Compute the arguments needed for the kernel launch
  ConvolutionSmallTileArgs<DIM> args;
  size_t blocks        = 1;
  size_t tile_pitch    = 1;
  unsigned input_pitch = 1;
  args.filter_volume   = 1;
  for (int d = DIM - 1; d >= 0; d--) {
    size_t blocks_along_dim = ((subrect.hi[d] - subrect.lo[d]) + tile[d]) / tile[d];
    args.grid_pitches[d]    = FastDivmodU64(blocks);
    blocks *= blocks_along_dim;
    args.block_tiles[d]   = tile[d];
    args.block_pitches[d] = FastDivmodU64(tile_pitch);
    tile_pitch *= tile[d];
    args.delta_lo[d]      = centers[d];
    args.delta_hi[d]      = tile[d] + centers[d] - 1;
    args.input_pitches[d] = FastDivmodU64(input_pitch);
    input_pitch *= (args.delta_lo[d] + args.delta_hi[d] + 1);
    args.filter_centers[d] = centers[d];
    args.filter_extents[d] = extents[d];
    args.filter_volume *= extents[d];
  }
  args.tile_volume  = tile_pitch;
  args.input_volume = input_pitch;
  assert((input_pitch * sizeof(VAL)) == smem_size);
  auto stream = get_cached_stream();
  if (halved) {
    if (tile_pitch < 512)
      convolution_small_tile1<VAL, DIM><<<blocks, tile_pitch, smem_size, stream>>>(
        out, filter, in, root_rect, subrect, filter_rect, args);
    else
      convolution_small_tile1<VAL, DIM><<<blocks, 512, smem_size, stream>>>(
        out, filter, in, root_rect, subrect, filter_rect, args);
  } else {
    if (tile_pitch < 1024)
      convolution_small_tile2<VAL, DIM><<<blocks, tile_pitch, smem_size, stream>>>(
        out, filter, in, root_rect, subrect, filter_rect, args);
    else
      convolution_small_tile2<VAL, DIM><<<blocks, 1024, smem_size, stream>>>(
        out, filter, in, root_rect, subrect, filter_rect, args);
  }
  CHECK_CUDA_STREAM(stream);
}

template <typename VAL, int32_t DIM>
__host__ void direct_convolution(AccessorWO<VAL, DIM> out,
                                 AccessorRO<VAL, DIM> filter,
                                 AccessorRO<VAL, DIM> in,
                                 const Rect<DIM>& root_rect,
                                 const Rect<DIM>& subrect,
                                 const Rect<DIM>& filter_rect)
{
  constexpr int THREADVALS = THREAD_OUTPUTS(VAL);
  // Get the maximum amount of shared memory per threadblock
  int device;
  CHECK_CUDA(hipGetDevice(&device));
  hipDeviceProp_t properties;
  CHECK_CUDA(hipGetDeviceProperties(&properties, device));
  size_t max_smem_size = properties.sharedMemPerBlockOptin;

  // Only need to do these calls the first time on each device so
  // we use a bit mask to track which devices we've done it for
  static unsigned long long mask = 0;
  if (!(mask & (1 << device))) {
    if (properties.sharedMemPerBlock < max_smem_size) {
      CHECK_CUDA(hipFuncSetAttribute(reinterpret_cast<const void*>(convolution_small_tile1<VAL), DIM>,
                                      hipFuncAttributeMaxDynamicSharedMemorySize,
                                      max_smem_size));
      CHECK_CUDA(hipFuncSetAttribute(reinterpret_cast<const void*>(convolution_small_tile2<VAL), DIM>,
                                      hipFuncAttributeMaxDynamicSharedMemorySize,
                                      max_smem_size));
      CHECK_CUDA(hipFuncSetAttribute(reinterpret_cast<const void*>(convolution_large_tile<VAL), DIM, THREADVALS>,
                                      hipFuncAttributeMaxDynamicSharedMemorySize,
                                      max_smem_size));
    }
    if (sizeof(VAL) >= 8) {
      // Only need to set this on the first invocation
      CHECK_CUDA(hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(convolution_small_tile1<VAL), DIM>,
                                            hipSharedMemBankSizeEightByte));
      CHECK_CUDA(hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(convolution_small_tile2<VAL), DIM>,
                                            hipSharedMemBankSizeEightByte));
      CHECK_CUDA(hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(convolution_large_tile<VAL), DIM, THREADVALS>,
                                            hipSharedMemBankSizeEightByte));
    }
    // Make sure we have enough bits for every device
    assert(device < (8 * sizeof(mask)));
    // Make sure not to race with updates from other GPUs
    __sync_fetch_and_add(&mask, (1 << device));
  }
  unsigned extents[DIM];
  unsigned centers[DIM];
  for (int d = 0; d < DIM; d++) {
    assert(filter_rect.lo[d] == 0);
    extents[d] = filter_rect.hi[d] + 1;
    centers[d] = static_cast<coord_t>(extents[d] / 2);
  }
  Point<DIM> tile;
  for (int d = DIM - 1; d >= 0; d--) {
    // Make sure that each tile is at least double the size of the filter
    // so that we can get some savings in bandwidth needed
    tile[d] = 2 * centers[d];
    if (d == (DIM - 1)) {
      // In order to maximize bandwidth, we want to make sure we're loading at
      // least 128B of contiguous memory along the last axis (row-major) of input
      const unsigned min_contig_elmts = 128 / sizeof(VAL);
      if ((tile[d] + 2 * centers[d]) < min_contig_elmts)
        tile[d] = min_contig_elmts - 2 * centers[d];
    }
  }
  unsigned smem_size = sizeof(VAL);
  for (int d = 0; d < DIM; d++) smem_size *= (tile[d] + 2 * centers[d]);
  if (smem_size <= max_smem_size) {
    // Small tile case:
    launch_small_tile_kernel<VAL, DIM>(out,
                                       filter,
                                       in,
                                       root_rect,
                                       subrect,
                                       filter_rect,
                                       properties,
                                       extents,
                                       centers,
                                       tile,
                                       smem_size,
                                       max_smem_size);
  } else {
    // Large tile case:
    // If we're going to do this, we need to initialize the output to zeros
    // so we can kick that off to the GPU while we figure out how to launch
    // the rest of the kernels to do the convolution
    size_t strides[DIM];
    VAL* out_ptr = out.ptr(subrect, strides);
    // Check to see if the output is dense
    bool out_dense   = true;
    size_t out_pitch = 1;
    for (int d = DIM - 1; d >= 0; d--) {
      if (strides[d] != out_pitch) {
        out_dense = false;
        break;
      }
      out_pitch *= strides[d];
    }
    if (out_dense) {
      size_t bytes = sizeof(VAL) * out_pitch;
      CHECK_CUDA(hipMemsetAsync(out_ptr, 0, bytes));
    } else {
      out_pitch = 1;
      ConvolutionInitArgs<DIM> args;
      for (int d = DIM - 1; d >= 0; d--) {
        args.pitches[d] = FastDivmodU64(out_pitch);
        out_pitch *= (subrect.hi[d] - subrect.lo[d] + 1);
      }
      size_t blocks = (out_pitch + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
      convolution_init<VAL, DIM><<<blocks, THREADS_PER_BLOCK>>>(out, subrect.lo, args, out_pitch);
    }
    // Figure out the shape of the L1 output tile based on the number of
    // points that we can fit into registers
    Point<DIM> l1_output_tile;
    const unsigned max_l1_output_volume = CONVOLUTION_THREADS * THREADVALS;
    // Make sure the max_l1_output_volume doesn't consume more than half of shared memory
    unsigned target_l1_output_volume = max_l1_output_volume;
    while ((max_smem_size / 2) < (target_l1_output_volume * sizeof(VAL)))
      target_l1_output_volume /= 2;
    const Point<DIM> output_bounds = subrect.hi - subrect.lo + Point<DIM>::ONES();
    const unsigned l1_output_volume =
      compute_output_tile<VAL, DIM>(l1_output_tile,
                                    output_bounds,
                                    128 /*cache line size*/ / sizeof(VAL),
                                    target_l1_output_volume);
    // At this point we've got our output tile, compute how big a filter
    // tile we can make and still fit both the filter tile and the
    // input tile into the maximum amount of shared memory for this GPU
    Point<DIM> l1_filter_tile;
    const Point<DIM> filter_bounds = filter_rect.hi - filter_rect.lo + Point<DIM>::ONES();
    unsigned dynamic_smem =
      compute_filter_tile<VAL, DIM>(l1_filter_tile, filter_bounds, l1_output_tile, max_smem_size);
    unsigned input_smem_offset = 1;
    for (int d = 0; d < DIM; d++) input_smem_offset *= l1_filter_tile[d];
    // Tile the number of SMs on this GPU to compute the shape of the
    // L2 output tile for this kernel
    // We assume here that the number of SMs is easily factorable
    // into primes of 2, 3, and 5. It would be strange if we have a
    // GPU with a number of SMs these days that can't be factored
    // this way. If we do report a warning.
    unsigned l2_tiles[DIM];
    for (int d = 0; d < DIM; d++) l2_tiles[d] = 1;
    if (DIM > 1) {
      unsigned twos = 0, threes = 0, fives = 0;
      unsigned remainder = properties.multiProcessorCount;
      while ((remainder > 1) && ((remainder % 2) == 0)) {
        twos++;
        remainder /= 2;
      }
      while ((remainder > 1) && ((remainder % 3) == 0)) {
        threes++;
        remainder /= 3;
      }
      while ((remainder > 1) && ((remainder % 5) == 0)) {
        fives++;
        remainder /= 5;
      }
      if (remainder > 1) {
        fprintf(stdout,
                "WARNING: %d is an unusual number of SMs "
                "for GPU convolution. Please report your GPU kind and "
                "the number of SMs in a Legate NumPy issue.",
                properties.multiProcessorCount);
        l2_tiles[DIM - 1] = remainder;
      }
      for (unsigned idx = 0; idx < fives; idx++) {
        int smallest = 0;
        for (int d = 1; d < DIM; d++) {
          if (l2_tiles[smallest] < l2_tiles[d]) continue;
          smallest = d;
        }
        l2_tiles[smallest] *= 5;
      }
      for (unsigned idx = 0; idx < threes; idx++) {
        int smallest = 0;
        for (int d = 1; d < DIM; d++) {
          if (l2_tiles[smallest] < l2_tiles[d]) continue;
          smallest = d;
        }
        l2_tiles[smallest] *= 3;
      }
      for (unsigned idx = 0; idx < twos; idx++) {
        int smallest = 0;
        for (int d = 1; d < DIM; d++) {
          if (l2_tiles[smallest] < l2_tiles[d]) continue;
          smallest = d;
        }
        l2_tiles[smallest] *= 2;
      }
    } else {
      l2_tiles[0] = properties.multiProcessorCount;
    }
    // Now that we've got a tiling of the l1 output blocks across
    // the SMs compute how big it is in memory and see if it is less
    // than a quarter of the L2 cache so we can block for the L2
    Point<DIM> l2_output_tile;
    size_t l2_output_tile_size = sizeof(VAL);
    for (int d = 0; d < DIM; d++) {
      l2_output_tile[d] = l2_tiles[d] * l1_output_tile[d];
      l2_output_tile_size *= l2_output_tile[d];
    }
    Point<DIM> l2_filter_tile;
    size_t total_l2_filters = 1;
    if (l2_output_tile_size <= (properties.l2CacheSize / 4)) {
      for (int d = 0; d < DIM; d++) l2_filter_tile[d] = 1;
      // Compute the L2 filter tile size so that the L2 filter and the
      // corresponding L2 input tile will fit in the L2 cache
      compute_filter_tile<VAL, DIM>(
        l2_filter_tile, filter_bounds, l2_output_tile, 3 * properties.l2CacheSize / 4);
      for (int d = 0; d < DIM; d++)
        total_l2_filters *= (filter_bounds[d] + l2_filter_tile[d] - 1) / l2_filter_tile[d];
    } else {
      // It's likely this tile is too big to block for the L2 cache
      // so we're not going to bother blocking for the L2 and just
      // run everything out of the framebuffer memory. The upside is
      // that we'll only need to make a single pass over the input
      for (int d = 0; d < DIM; d++) l2_filter_tile[d] = filter_rect.hi[d] - filter_rect.lo[d] + 1;
    }
    // Construct the arguments for the kernel launches
    ConvolutionLargeTileArgs<DIM, THREADVALS> args;
    int pitch = 1;
    for (int d = DIM - 1; d >= 0; d--) {
      args.l1_input_pitches[d] = FastDivmod(pitch);
      pitch *= (l1_output_tile[d] + 2 * (l1_filter_tile[d] / 2));
    }
    pitch = 1;
    for (int d = DIM - 1; d >= 0; d--) {
      args.l1_filter_pitches[d] = FastDivmod(pitch);
      pitch *= l1_filter_tile[d];
    }
    pitch = 1;
    for (int d = DIM - 1; d >= 0; d--) {
      args.l1_output_pitches[d] = FastDivmod(pitch);
      pitch *= l1_output_tile[d];
    }
    args.l2_output_tile      = l2_output_tile;
    args.l2_filter_tile      = l2_filter_tile;
    args.l1_output_tile      = l1_output_tile;
    args.l1_filter_tile      = l1_filter_tile;
    args.l2_output_limits    = output_bounds;
    args.shared_input_offset = input_smem_offset;
    args.total_l2_outputs    = 1;
    args.total_l1_outputs    = 1;
    args.total_l1_filters    = 1;
    args.l1_filter_points    = 1;
    args.l1_input_points     = 1;
    pitch                    = 1;
    for (int d = DIM - 1; d >= 0; d--) {
      args.total_l2_outputs *= (output_bounds[d] + l2_output_tile[d] - 1) / l2_output_tile[d];
      args.l1_output_tile_pitches[d] = FastDivmod(pitch);
      pitch *= (l2_output_tile[d] + l1_output_tile[d] - 1) / l1_output_tile[d];
      args.total_l1_filters *= (l2_filter_tile[d] + l1_filter_tile[d] - 1) / l1_filter_tile[d];
      args.l1_filter_points *= l1_filter_tile[d];
      args.l1_input_points *= (l1_output_tile[d] + 2 * (l1_filter_tile[d] / 2));
    }
    args.total_l1_outputs = pitch;
    // Figure out how to tile the points across the l1_output_tile
    if (DIM > 1) {
      unsigned regsteps[DIM];
      for (int d = 0; d < DIM; d++) regsteps[d] = 0;
      unsigned remainder = THREADVALS;
      // Handle the case here where we aren't going to use all
      // the points in the registers so we need to scale back
      if (l1_output_volume < max_l1_output_volume) {
        assert((max_l1_output_volume % l1_output_volume) == 0);
        remainder /= (max_l1_output_volume / l1_output_volume);
        if (remainder == 0) remainder = 1;
      }
      for (int d = 0; d < DIM; d++) {
        if (remainder == 1) {
          regsteps[d] = l1_output_tile[d];
        } else if (remainder <= l1_output_tile[d]) {
          // All powers of two so should always divide
          assert((l1_output_tile[d] % remainder) == 0);
          regsteps[d] = l1_output_tile[d] / remainder;
          remainder   = 1;
        } else {
          // All powers of two so should always divide
          assert((remainder % l1_output_tile[d]) == 0);
          regsteps[d] = 1;
          remainder /= l1_output_tile[d];
        }
      }
      assert(remainder == 1);
      Point<DIM, unsigned> offset = Point<DIM, unsigned>::ZEROES();
      for (int p = 0; p < THREADVALS; p++) {
        args.point_offsets[p] = offset;
        // Step to the next offset
        for (int d = DIM - 1; d >= 0; d--) {
          offset[d] += regsteps[d];
          if (offset[d] == l1_output_tile[d]) {
            if ((d == 0) && (p != (THREADVALS - 1)))
              // Allow overflow in this case to handle the case
              // where we have more points than we need for the l1 output tile
              assert(l1_output_volume < max_l1_output_volume);
            else
              offset[d] = 0;
          } else
            break;
        }
      }
      args.uniform_input_stride = regsteps[0] * args.l1_input_pitches[0].divisor;
      // Check to make sure this is the uniform input stride case
      for (int d = 1; d < DIM; d++) {
        if (regsteps[d] == l1_output_tile[d]) continue;
        args.uniform_input_stride = 0;
        break;
      }
    } else {
      assert(THREADVALS <= l1_output_tile[0]);
      unsigned remainder = THREADVALS;
      // Handle the case here where we aren't going to use all
      // the points in the registers so we need to scale back
      if (l1_output_volume < max_l1_output_volume) {
        assert((max_l1_output_volume % l1_output_volume) == 0);
        remainder /= (max_l1_output_volume / l1_output_volume);
        if (remainder == 0) remainder = 1;
      }
      assert((l1_output_tile[0] % remainder) == 0);
      unsigned regstep = l1_output_tile[0] / remainder;
      for (int p = 0; p < THREADVALS; p++) args.point_offsets[p][0] = p * regstep;
      args.uniform_input_stride = regstep * args.l1_input_pitches[0].divisor;
    }
    if (l1_output_volume < max_l1_output_volume) {
      args.shared_input_bound = dynamic_smem / sizeof(VAL);
      args.total_l1_points    = l1_output_volume;
    } else {
      args.shared_input_bound = 0;
      args.total_l1_points    = 0;
    }
    // Launch as many kernels as we need to walk over the entire filter
    // Given the L2 filter tile that we came up with
    auto stream                     = get_cached_stream();
    const Point<DIM, unsigned> zero = Point<DIM, unsigned>::ZEROES();
    const Point<DIM, unsigned> one  = Point<DIM, unsigned>::ONES();
    if (total_l2_filters > 1) {
      Point<DIM> l2_filter_lo = filter_rect.lo;
      for (unsigned idx = 0; idx < total_l2_filters; idx++) {
        Rect<DIM> l2_filter_rect(l2_filter_lo, l2_filter_lo + l2_filter_tile - one);
        l2_filter_rect = l2_filter_rect.intersection(filter_rect);
        const Point<DIM> l1_input_start =
          subrect.lo + Point<DIM>(extents) - l2_filter_lo - l1_filter_tile - Point<DIM>(centers);
        const Point<DIM> l2_input_start =
          subrect.lo + Point<DIM>(extents) - l2_filter_rect.hi - one - Point<DIM>(centers);
        const Point<DIM> l2_input_stop = subrect.lo + l2_output_tile - one + Point<DIM>(extents) -
                                         l2_filter_rect.lo - one - Point<DIM>(centers);
        convolution_large_tile<VAL, DIM, THREADVALS>
          <<<properties.multiProcessorCount, CONVOLUTION_THREADS, dynamic_smem, stream>>>(
            out,
            filter,
            in,
            root_rect,
            subrect,
            l2_filter_rect,
            l2_input_start,
            l2_input_stop,
            l1_input_start,
            zero,
            one,
            args);
        // Step to the next filter
        for (int d = DIM - 1; d >= 0; d--) {
          l2_filter_lo[d] += l2_filter_tile[d];
          if (filter_rect.hi[d] < l2_filter_lo[d])
            l2_filter_lo[d] = filter_rect.lo[d];
          else
            break;
        }
      }
    } else {
      assert(total_l2_filters == 1);
      const Point<DIM> l1_input_start =
        subrect.lo + Point<DIM>(extents) - filter_rect.lo - l1_filter_tile - Point<DIM>(centers);
      const Point<DIM> l2_input_start = subrect.lo - Point<DIM>(centers);
      const Point<DIM> l2_input_stop  = subrect.lo + l2_output_tile - one + Point<DIM>(extents) -
                                       filter_rect.lo - one - Point<DIM>(centers);
      convolution_large_tile<VAL, DIM, THREADVALS>
        <<<properties.multiProcessorCount, CONVOLUTION_THREADS, dynamic_smem, stream>>>(
          out,
          filter,
          in,
          root_rect,
          subrect,
          filter_rect,
          l2_input_start,
          l2_input_stop,
          l1_input_start,
          zero,
          one,
          args);
    }
    CHECK_CUDA_STREAM(stream);
  }
}

///////////////////////////////////////
// FFT-based convolution implementation
///////////////////////////////////////

template <int DIM>
struct FFTPitches {
  size_t pitches[DIM];
  __host__ inline size_t& operator[](unsigned idx) { return pitches[idx]; }
  __device__ __forceinline__ size_t operator[](unsigned idx) const { return pitches[idx]; }
};

template <int DIM>
struct CopyPitches {
  FastDivmodU64 pitches[DIM];
  __host__ inline FastDivmodU64& operator[](unsigned idx) { return pitches[idx]; }
  __device__ __forceinline__ const FastDivmodU64& operator[](unsigned idx) const
  {
    return pitches[idx];
  }
};

template <typename VAL, int DIM>
__global__ static void __launch_bounds__(THREADS_PER_BLOCK, 4)
  copy_into_buffer(const AccessorRO<VAL, DIM> accessor,
                   const Buffer<VAL, DIM> buffer,
                   const Point<DIM> lo,
                   const CopyPitches<DIM> copy_pitches,
                   const size_t volume)
{
  size_t offset = blockIdx.x * blockDim.x + threadIdx.x;
  if (offset >= volume) return;
  Point<DIM> point;
  for (int d = 0; d < DIM; d++) point[d] = copy_pitches[d].divmod(offset, offset);
  buffer[point] = accessor[lo + point];
}

template <typename VAL, int DIM>
__global__ static void __launch_bounds__(THREADS_PER_BLOCK, 4)
  copy_from_buffer(const VAL* buffer,
                   const AccessorWO<VAL, DIM> accessor,
                   const Point<DIM> buffer_lo,
                   const Point<DIM> accessor_lo,
                   const CopyPitches<DIM> copy_pitches,
                   const FFTPitches<DIM> fft_pitches,
                   const size_t volume,
                   const VAL scaling)
{
  size_t offset = blockIdx.x * blockDim.x + threadIdx.x;
  if (offset >= volume) return;
  Point<DIM> point;
  size_t buffer_offset = 0;
  for (int d = 0; d < DIM; d++) {
    point[d] = copy_pitches[d].divmod(offset, offset);
    buffer_offset += (buffer_lo[d] + point[d]) * fft_pitches[d];
  }
  accessor[accessor_lo + point] = scaling * buffer[buffer_offset];
}

template <typename VAL>
__global__ static void __launch_bounds__(THREADS_PER_BLOCK, 4)
  complex_multiply(complex<VAL>* inout, complex<VAL>* in, const size_t volume)
{
  size_t offset = blockIdx.x * blockDim.x + threadIdx.x;
  if (offset >= volume) return;
  inout[offset] *= in[offset];
}

__host__ inline void cufft_execute_forward(hipfftHandle plan, float* idata, float* odata)
{
  CHECK_CUFFT(hipfftExecR2C(plan, (hipfftReal*)idata, (hipfftComplex*)odata));
}

__host__ inline void cufft_execute_forward(hipfftHandle plan, double* idata, double* odata)
{
  CHECK_CUFFT(hipfftExecD2Z(plan, (hipfftDoubleReal*)idata, (hipfftDoubleComplex*)odata));
}

__host__ inline void cufft_execute_backward(hipfftHandle plan, float* idata, float* odata)
{
  CHECK_CUFFT(hipfftExecC2R(plan, (hipfftComplex*)idata, (hipfftReal*)odata));
}

__host__ inline void cufft_execute_backward(hipfftHandle plan, double* idata, double* odata)
{
  CHECK_CUFFT(hipfftExecZ2D(plan, (hipfftDoubleComplex*)idata, (hipfftDoubleReal*)odata));
}

template <typename VAL>
struct ForwardPlanType;

template <>
struct ForwardPlanType<float> {
  static constexpr hipfftType value = HIPFFT_R2C;
};

template <>
struct ForwardPlanType<double> {
  static constexpr hipfftType value = HIPFFT_D2Z;
};

template <typename VAL>
struct BackwardPlanType;

template <>
struct BackwardPlanType<float> {
  static constexpr hipfftType value = HIPFFT_C2R;
};

template <>
struct BackwardPlanType<double> {
  static constexpr hipfftType value = HIPFFT_Z2D;
};

template <typename VAL, int DIM>
__host__ static inline void cufft_convolution(AccessorWO<VAL, DIM> out,
                                              AccessorRO<VAL, DIM> filter,
                                              AccessorRO<VAL, DIM> in,
                                              const Rect<DIM>& root_rect,
                                              const Rect<DIM>& subrect,
                                              const Rect<DIM>& filter_rect)
{
  int device;
  CHECK_CUDA(hipGetDevice(&device));
  hipDeviceProp_t properties;
  CHECK_CUDA(hipGetDeviceProperties(&properties, device));
  size_t max_smem_size = properties.sharedMemPerBlockOptin;

  // Only need to do these calls the first time on each device so
  // we use a bit mask to track which devices we've done it for
  static unsigned long long mask = 0;
  if (!(mask & (1 << device))) {
    if (properties.sharedMemPerBlock < max_smem_size) {
      CHECK_CUDA(hipFuncSetAttribute(reinterpret_cast<const void*>(convolution_small_tile1<VAL), DIM>,
                                      hipFuncAttributeMaxDynamicSharedMemorySize,
                                      max_smem_size));
      CHECK_CUDA(hipFuncSetAttribute(reinterpret_cast<const void*>(convolution_small_tile2<VAL), DIM>,
                                      hipFuncAttributeMaxDynamicSharedMemorySize,
                                      max_smem_size));
    }
    if (sizeof(VAL) >= 8) {
      // Only need to set this on the first invocation
      CHECK_CUDA(hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(convolution_small_tile1<VAL), DIM>,
                                            hipSharedMemBankSizeEightByte));
      CHECK_CUDA(hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(convolution_small_tile2<VAL), DIM>,
                                            hipSharedMemBankSizeEightByte));
    }
    // Make sure we have enough bits for every device
    assert(device < (8 * sizeof(mask)));
    // Make sure not to race with updates from other GPUs
    __sync_fetch_and_add(&mask, (1 << device));
  }
  unsigned extents[DIM];
  unsigned centers[DIM];
  for (int d = 0; d < DIM; d++) {
    assert(filter_rect.lo[d] == 0);
    extents[d] = filter_rect.hi[d] + 1;
    centers[d] = static_cast<coord_t>(extents[d] / 2);
  }
  Point<DIM> tile;
  for (int d = DIM - 1; d >= 0; d--) {
    // Make sure that each tile is at least double the size of the filter
    // so that we can get some savings in bandwidth needed
    tile[d] = 2 * centers[d];
    if (d == (DIM - 1)) {
      // In order to maximize bandwidth, we want to make sure we're loading at
      // least 128B of contiguous memory along the last axis (row-major) of input
      const unsigned min_contig_elmts = 128 / sizeof(VAL);
      if ((tile[d] + 2 * centers[d]) < min_contig_elmts)
        tile[d] = min_contig_elmts - 2 * centers[d];
    }
  }
  unsigned smem_size = sizeof(VAL);
  for (int d = 0; d < DIM; d++) smem_size *= (tile[d] + 2 * centers[d]);
  if (smem_size <= max_smem_size) {
    launch_small_tile_kernel<VAL, DIM>(out,
                                       filter,
                                       in,
                                       root_rect,
                                       subrect,
                                       filter_rect,
                                       properties,
                                       extents,
                                       centers,
                                       tile,
                                       smem_size,
                                       max_smem_size);
  } else {
    // Instead of doing the large tile case, we can instead do this
    // by transforming both the input and the filter to the frequency
    // domain using an FFT, perform the convolution with a point-wise
    // multiplication, and then transform the result back to the spatial domain
    auto stream = get_cached_stream();
    // First compute how big our temporary allocation needs to be
    // We'll need two of them to store the zero-padded data for the inputs
    const Point<DIM> zero = Point<DIM>::ZEROES();
    const Point<DIM> one  = Point<DIM>::ONES();
    Rect<DIM> offset_bounds;
    for (int d = 0; d < DIM; d++) {
      offset_bounds.lo[d] = subrect.lo[d] - centers[d];
      offset_bounds.hi[d] = subrect.hi[d] + extents[d] - 1 - centers[d];
    }
    Rect<DIM> input_bounds         = root_rect.intersection(offset_bounds);
    const Point<DIM> signal_bounds = input_bounds.hi - input_bounds.lo + one;
    const Point<DIM> filter_bounds = filter_rect.hi - filter_rect.lo + one;
    Point<DIM> fftsize             = signal_bounds + filter_bounds;
    for (int d = 0; d < DIM; d++) {
      // Technically we can shrink this by one and still be sound but we'll
      // only do that if it will make the number even
      if ((fftsize[d] % 2) == 1) fftsize[d]--;
    }
    // Cufft needs the last dimension to have fftsize/2+1 complex elements for
    // the temporary buffer
    // Since we know fftsize is even, we just need to add two to it for the output
    Point<DIM> buffersize = fftsize;
    buffersize[DIM - 1] += 2;
    size_t buffervolume = 1;
    for (int d = 0; d < DIM; d++) buffervolume *= buffersize[d];
    // Zero pad and copy in the input data
    auto signal_buffer = create_buffer<VAL, DIM>(buffersize, Memory::GPU_FB_MEM, 128 /*alignment*/);
    VAL* signal_ptr    = signal_buffer.ptr(zero);
    CHECK_CUDA(hipMemsetAsync(signal_ptr, 0, buffervolume * sizeof(VAL), stream));
    // Check to see if the input pointer is dense and we can do this with a CUDA memcpy
    size_t strides[DIM];
    const VAL* input_ptr = in.ptr(input_bounds, strides);
    size_t pitch         = 1;
    CopyPitches<DIM> copy_pitches;
    for (int d = DIM - 1; d >= 0; d--) {
      copy_pitches[d] = FastDivmodU64(pitch);
      pitch *= signal_bounds[d];
    }
    size_t blocks = (pitch + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    copy_into_buffer<VAL, DIM><<<blocks, THREADS_PER_BLOCK, 0, stream>>>(
      in, signal_buffer, input_bounds.lo, copy_pitches, pitch);
    // Zero pad and copy in the filter data
    auto filter_buffer = create_buffer<VAL, DIM>(buffersize, Memory::GPU_FB_MEM, 128 /*alignment*/);
    VAL* filter_ptr    = filter_buffer.ptr(zero);
    CHECK_CUDA(hipMemsetAsync(filter_ptr, 0, buffervolume * sizeof(VAL), stream));
    const VAL* filt_ptr = filter.ptr(filter_rect, strides);
    pitch               = 1;
    for (int d = DIM - 1; d >= 0; d--) {
      copy_pitches[d] = FastDivmodU64(pitch);
      pitch *= filter_bounds[d];
    }
    blocks = (pitch + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    copy_into_buffer<VAL, DIM><<<blocks, THREADS_PER_BLOCK, 0, stream>>>(
      filter, filter_buffer, filter_rect.lo, copy_pitches, pitch);

    CHECK_CUDA_STREAM(stream);

    auto forward_plan  = get_cufft_plan(ForwardPlanType<VAL>::value, fftsize);
    auto backward_plan = get_cufft_plan(BackwardPlanType<VAL>::value, fftsize);

    // Set the stream and working area for the plans
    CHECK_CUFFT(hipfftSetStream(forward_plan.handle(), stream));
    CHECK_CUFFT(hipfftSetStream(backward_plan.handle(), stream));

    auto workarea_size = std::max(forward_plan.workareaSize(), backward_plan.workareaSize());

    // Create the plan and allocate a temporary buffer for it if it needs one
    Buffer<uint8_t, 1> workarea_buffer;
    if (workarea_size > 0) {
      const Point<1> zero1d(0);
      workarea_buffer =
        create_buffer<uint8_t, 1>(workarea_size, Memory::GPU_FB_MEM, 128 /*alignment*/);
      void* workarea = workarea_buffer.ptr(zero1d);
      CHECK_CUFFT(hipfftSetWorkArea(forward_plan.handle(), workarea));
      CHECK_CUFFT(hipfftSetWorkArea(backward_plan.handle(), workarea));
    }
    // FFT the input data
    cufft_execute_forward(forward_plan.handle(), signal_ptr, signal_ptr);
    // FFT the filter data
    cufft_execute_forward(forward_plan.handle(), filter_ptr, filter_ptr);

    CHECK_CUDA_STREAM(stream);

    // Perform the pointwise multiplcation
    {
      size_t volume = (buffervolume / 2);
      blocks        = (volume + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
      complex_multiply<VAL><<<blocks, THREADS_PER_BLOCK, 0, stream>>>(
        (complex<VAL>*)signal_ptr, (complex<VAL>*)filter_ptr, volume);
    }
    // Inverse FFT for the ouptut
    // Allow this out-of-place for better performance
    cufft_execute_backward(backward_plan.handle(), signal_ptr, filter_ptr);
    // Copy the result data out of the temporary buffer and scale
    // because CUFFT inverse does not perform the scale for us
    pitch = 1;
    FFTPitches<DIM> fft_pitches;
    for (int d = DIM - 1; d >= 0; d--) {
      fft_pitches[d] = pitch;
      pitch *= fftsize[d];
    }
    const VAL scaling_factor = VAL(1) / pitch;
    Point<DIM> buffer_offset;
    for (int d = 0; d < DIM; d++)
      buffer_offset[d] =
        centers[d] - (((extents[d] % 2) == 0) ? 1 : 0) +
        ((offset_bounds.lo[d] < root_rect.lo[d]) ? (subrect.lo[d] - root_rect.lo[d]) : centers[d]);
    Point<DIM> output_bounds = subrect.hi - subrect.lo + one;
    pitch                    = 1;
    for (int d = DIM - 1; d >= 0; d--) {
      copy_pitches[d] = FastDivmodU64(pitch);
      pitch *= output_bounds[d];
    }
    blocks = (pitch + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    copy_from_buffer<VAL, DIM><<<blocks, THREADS_PER_BLOCK, 0, stream>>>(
      filter_ptr, out, buffer_offset, subrect.lo, copy_pitches, fft_pitches, pitch, scaling_factor);

    CHECK_CUDA_STREAM(stream);

#if 0
    // This is useful debugging code for finding the output
    VAL *buffer = (VAL*)malloc(buffervolume*sizeof(VAL));
    CHECK_CUDA( hipMemcpyAsync(buffer, filter_ptr, buffervolume*sizeof(VAL), hipMemcpyDeviceToHost, stream) );
    CHECK_CUDA( hipStreamSynchronize(stream) );
    for (unsigned idx = 0; idx < buffervolume; idx++) {
      if ((idx % fftsize[DIM-1]) == 0)
        printf("\n");
      printf("%.8g ", buffer[idx]*scaling_factor);
    }
    printf("\n");
    free(buffer);
#endif
  }
}

/////////////
// Dispatcher
/////////////

template <typename VAL, int DIM>
struct UseCUFFT {
  static constexpr bool value = 1 <= DIM && DIM <= 3 && std::is_floating_point<VAL>::value;
};

template <Type CODE, int DIM>
struct ConvolveImplBody<VariantKind::GPU, CODE, DIM> {
  using VAL = legate_type_of<CODE>;

  template <typename _VAL, int32_t _DIM, std::enable_if_t<UseCUFFT<_VAL, _DIM>::value>* = nullptr>
  __host__ void dispatch(AccessorWO<_VAL, _DIM> out,
                         AccessorRO<_VAL, _DIM> filter,
                         AccessorRO<_VAL, _DIM> in,
                         const Rect<_DIM>& root_rect,
                         const Rect<_DIM>& subrect,
                         const Rect<_DIM>& filter_rect) const
  {
    cufft_convolution<_VAL, _DIM>(out, filter, in, root_rect, subrect, filter_rect);
  }

  template <typename _VAL, int32_t _DIM, std::enable_if_t<!UseCUFFT<_VAL, _DIM>::value>* = nullptr>
  __host__ void dispatch(AccessorWO<_VAL, _DIM> out,
                         AccessorRO<_VAL, _DIM> filter,
                         AccessorRO<_VAL, _DIM> in,
                         const Rect<_DIM>& root_rect,
                         const Rect<_DIM>& subrect,
                         const Rect<_DIM>& filter_rect) const
  {
    direct_convolution<_VAL, _DIM>(out, filter, in, root_rect, subrect, filter_rect);
  }

  __host__ void operator()(AccessorWO<VAL, DIM> out,
                           AccessorRO<VAL, DIM> filter,
                           AccessorRO<VAL, DIM> in,
                           const Rect<DIM>& root_rect,
                           const Rect<DIM>& subrect,
                           const Rect<DIM>& filter_rect) const
  {
    dispatch(out, filter, in, root_rect, subrect, filter_rect);
  }
};

/*static*/ void ConvolveTask::gpu_variant(TaskContext& context)
{
  convolve_template<VariantKind::GPU>(context);
}

}  // namespace cunumeric
