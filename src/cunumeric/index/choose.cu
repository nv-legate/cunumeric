#include "hip/hip_runtime.h"
/* Copyright 2021-2022 NVIDIA Corporation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */

#include "cunumeric/index/choose.h"
#include "cunumeric/index/choose_template.inl"
#include "cunumeric/cuda_help.h"

namespace cunumeric {

using namespace Legion;

template <typename VAL, int DIM>
__global__ static void __launch_bounds__(THREADS_PER_BLOCK, MIN_CTAS_PER_SM)
  choose_kernel(const AccessorWO<VAL, DIM> out,
                const AccessorRO<int64_t, DIM> index_arr,
                const Buffer<AccessorRO<VAL, DIM>, 1> choices,
                const Rect<DIM> rect,
                const Pitches<DIM - 1> pitches,
                int volume)
{
  const size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= volume) return;
  auto p = pitches.unflatten(idx, rect.lo);
  out[p] = choices[index_arr[p]][p];
}

// dense version
template <typename VAL>
__global__ static void __launch_bounds__(THREADS_PER_BLOCK, MIN_CTAS_PER_SM) choose_kernel_dense(
  VAL* outptr, const int64_t* indexptr, Buffer<const VAL*, 1> choices, int volume)
{
  const size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= volume) return;
  outptr[idx] = choices[indexptr[idx]][idx];
}

template <LegateTypeCode CODE, int DIM>
struct ChooseImplBody<VariantKind::GPU, CODE, DIM> {
  using VAL = legate_type_of<CODE>;

  void operator()(const AccessorWO<VAL, DIM>& out,
                  const AccessorRO<int64_t, DIM>& index_arr,
                  const std::vector<AccessorRO<VAL, DIM>>& choices,
                  const Rect<DIM>& rect,
                  const Pitches<DIM - 1>& pitches,
                  bool dense) const
  {
    const size_t volume = rect.volume();
    const size_t blocks = (volume + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    if (dense) {
      auto ch_arr = create_buffer<const VAL*>(choices.size(), Memory::Kind::Z_COPY_MEM);
      for (uint32_t idx = 0; idx < choices.size(); ++idx) ch_arr[idx] = choices[idx].ptr(rect);
      VAL* outptr             = out.ptr(rect);
      const int64_t* indexptr = index_arr.ptr(rect);
      choose_kernel_dense<VAL><<<blocks, THREADS_PER_BLOCK>>>(outptr, indexptr, ch_arr, volume);
    } else {
      auto ch_arr = create_buffer<AccessorRO<VAL, DIM>>(choices.size(), Memory::Kind::Z_COPY_MEM);
      for (uint32_t idx = 0; idx < choices.size(); ++idx) ch_arr[idx] = choices[idx];
      choose_kernel<VAL, DIM>
        <<<blocks, THREADS_PER_BLOCK>>>(out, index_arr, ch_arr, rect, pitches, volume);
    }
  }
};

/*static*/ void ChooseTask::gpu_variant(TaskContext& context)
{
  choose_template<VariantKind::GPU>(context);
}
}  // namespace cunumeric
