/* Copyright 2021-2022 NVIDIA Corporation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */

#include "cunumeric.h"

#include "cudalibs.h"

#include <stdio.h>

namespace cunumeric {

using namespace Legion;

static Logger log_cudalibs("cunumeric.cudalibs");

cufftContext::cufftContext(cufftPlan* plan) : plan_(plan) {}

cufftContext::~cufftContext()
{
  auto hdl = handle();
  for (auto type : callback_types_) CHECK_CUFFT(hipfftXtClearCallback(hdl, type));
}

hipfftHandle cufftContext::handle() { return plan_->handle; }

size_t cufftContext::workareaSize() { return plan_->workarea_size; }

void cufftContext::setCallback(hipfftXtCallbackType type, void* callback, void* data)
{
  void* callbacks[] = {callback};
  void* datas[]     = {data};
  CHECK_CUFFT(hipfftXtSetCallback(handle(), callbacks, type, datas));
}

struct cufftPlanCache {
 private:
  // Maximum number of plans to keep per dimension
  static constexpr int32_t MAX_PLANS = 4;

 private:
  struct LRUEntry {
    std::unique_ptr<cufftPlan> plan{nullptr};
    DomainPoint fftshape{};
    uint32_t lru_index{0};
  };

 public:
  cufftPlanCache(hipfftType type);
  ~cufftPlanCache();

 public:
  cufftPlan* get_cufft_plan(const DomainPoint& size);

 private:
  using Cache = std::array<LRUEntry, MAX_PLANS>;
  std::array<Cache, LEGION_MAX_DIM + 1> cache_{};
  hipfftType type_;
};

cufftPlanCache::cufftPlanCache(hipfftType type) : type_(type)
{
  for (auto& cache : cache_)
    for (auto& entry : cache) assert(0 == entry.fftshape.dim);
}

cufftPlanCache::~cufftPlanCache()
{
  for (auto& cache : cache_)
    for (auto& entry : cache)
      if (entry.plan != nullptr) CHECK_CUFFT(hipfftDestroy(entry.plan->handle));
}

cufftPlan* cufftPlanCache::get_cufft_plan(const DomainPoint& size)
{
  int32_t match = -1;
  auto& cache   = cache_[size.dim];
  for (int32_t idx = 0; idx < MAX_PLANS; ++idx)
    if (cache[idx].fftshape == size) {
      match = idx;
      break;
    }

  cufftPlan* result{nullptr};
  // If there's no match, we create a new plan
  if (-1 == match) {
    log_cudalibs.debug() << "[cufftPlanCache] no match found for " << size << " (type: " << type_
                         << ")";
    int32_t plan_index = -1;
    for (int32_t idx = 0; idx < MAX_PLANS; ++idx) {
      auto& entry = cache[idx];
      if (nullptr == entry.plan) {
        log_cudalibs.debug() << "[cufftPlanCache] found empty entry " << idx << " (type: " << type_
                             << ")";
        entry.plan      = std::make_unique<cufftPlan>();
        entry.lru_index = idx;
        plan_index      = idx;
        break;
      } else if (entry.lru_index == MAX_PLANS - 1) {
        log_cudalibs.debug() << "[cufftPlanCache] evict entry " << idx << " for " << entry.fftshape
                             << " (type: " << type_ << ")";
        CHECK_CUFFT(hipfftDestroy(entry.plan->handle));
        plan_index = idx;
        break;
      }
    }
    assert(plan_index != -1);
    auto& entry    = cache[plan_index];
    entry.fftshape = size;
    result         = entry.plan.get();

    CHECK_CUFFT(hipfftCreate(&result->handle));
    CHECK_CUFFT(hipfftSetAutoAllocation(result->handle, 0 /*we'll do the allocation*/));

    std::vector<int32_t> n(size.dim);
    for (int32_t dim = 0; dim < size.dim; ++dim) n[dim] = size[dim];
    CHECK_CUFFT(hipfftMakePlanMany(result->handle,
                                  size.dim,
                                  n.data(),
                                  nullptr,
                                  1,
                                  1,
                                  nullptr,
                                  1,
                                  1,
                                  type_,
                                  1 /*batch*/,
                                  &result->workarea_size));
  }
  // Otherwise, we return the cached plan and adjust the LRU count
  else {
    log_cudalibs.debug() << "[cufftPlanCache] found match for " << size << " (type: " << type_
                         << ")";
    auto& entry = cache[match];
    result      = entry.plan.get();

    for (int32_t idx = 0; idx < MAX_PLANS; ++idx) {
      auto& other = cache[idx];
      if (other.lru_index < entry.lru_index) ++other.lru_index;
    }
    entry.lru_index = 0;
  }
  return result;
}

CUDALibraries::CUDALibraries()
  : finalized_(false), cublas_(nullptr), cusolver_(nullptr), cutensor_(nullptr), plan_caches_()
{
}

CUDALibraries::~CUDALibraries() { finalize(); }

void CUDALibraries::finalize()
{
  if (finalized_) return;
  if (cublas_ != nullptr) finalize_cublas();
  if (cusolver_ != nullptr) finalize_cusolver();
  if (cutensor_ != nullptr) finalize_cutensor();
  for (auto& pair : plan_caches_) delete pair.second;
  finalized_ = true;
}

void CUDALibraries::finalize_cublas()
{
  CHECK_CUBLAS(hipblasDestroy(cublas_));
  cublas_ = nullptr;
}

void CUDALibraries::finalize_cusolver()
{
  CHECK_CUSOLVER(hipsolverDnDestroy(cusolver_));
  cusolver_ = nullptr;
}

void CUDALibraries::finalize_cutensor()
{
  delete cutensor_;
  cutensor_ = nullptr;
}

hipblasHandle_t CUDALibraries::get_cublas()
{
  if (nullptr == cublas_) {
    CHECK_CUBLAS(hipblasCreate(&cublas_));
    const char* fast_math = getenv("CUNUMERIC_FAST_MATH");
    if (fast_math != nullptr && atoi(fast_math) > 0) {
      // Enable acceleration of single precision routines using TF32 tensor cores.
      hipblasStatus_t status = hipblasSetMathMode(cublas_, HIPBLAS_TF32_TENSOR_OP_MATH);
      if (status != HIPBLAS_STATUS_SUCCESS)
        fprintf(stderr, "WARNING: cuBLAS does not support Tensor cores!");
    }
  }
  return cublas_;
}

hipsolverHandle_t CUDALibraries::get_cusolver()
{
  if (nullptr == cusolver_) CHECK_CUSOLVER(hipsolverDnCreate(&cusolver_));
  return cusolver_;
}

hiptensorHandle_t* CUDALibraries::get_cutensor()
{
  if (nullptr == cutensor_) {
    cutensor_ = new hiptensorHandle_t;
    CHECK_CUTENSOR(cutensorInit(cutensor_));
  }
  return cutensor_;
}

cufftContext CUDALibraries::get_cufft_plan(hipfftType type, const DomainPoint& size)
{
  auto finder = plan_caches_.find(type);
  cufftPlanCache* cache{nullptr};

  if (plan_caches_.end() == finder) {
    cache              = new cufftPlanCache(type);
    plan_caches_[type] = cache;
  } else
    cache = finder->second;
  return cufftContext(cache->get_cufft_plan(size));
}

static CUDALibraries& get_cuda_libraries(Processor proc)
{
  if (proc.kind() != Processor::TOC_PROC) {
    fprintf(stderr, "Illegal request for CUDA libraries for non-GPU processor");
    LEGATE_ABORT;
  }

  static CUDALibraries cuda_libraries[LEGION_MAX_NUM_PROCS];
  const auto proc_id = proc.id & (LEGION_MAX_NUM_PROCS - 1);
  return cuda_libraries[proc_id];
}

legate::cuda::StreamView get_cached_stream()
{
  return legate::cuda::StreamPool::get_stream_pool().get_stream();
}

cublasContext* get_cublas()
{
  const auto proc = Processor::get_executing_processor();
  auto& lib       = get_cuda_libraries(proc);
  return lib.get_cublas();
}

cusolverDnContext* get_cusolver()
{
  const auto proc = Processor::get_executing_processor();
  auto& lib       = get_cuda_libraries(proc);
  return lib.get_cusolver();
}

hiptensorHandle_t* get_cutensor()
{
  const auto proc = Processor::get_executing_processor();
  auto& lib       = get_cuda_libraries(proc);
  return lib.get_cutensor();
}

cufftContext get_cufft_plan(hipfftType type, const Legion::DomainPoint& size)
{
  const auto proc = Processor::get_executing_processor();
  auto& lib       = get_cuda_libraries(proc);
  return lib.get_cufft_plan(type, size);
}

class LoadCUDALibsTask : public CuNumericTask<LoadCUDALibsTask> {
 public:
  static const int TASK_ID = CUNUMERIC_LOAD_CUDALIBS;

 public:
  static void gpu_variant(legate::TaskContext& context)
  {
    const auto proc = Processor::get_executing_processor();
    auto& lib       = get_cuda_libraries(proc);
    lib.get_cublas();
    lib.get_cusolver();
    lib.get_cutensor();
  }
};

class UnloadCUDALibsTask : public CuNumericTask<UnloadCUDALibsTask> {
 public:
  static const int TASK_ID = CUNUMERIC_UNLOAD_CUDALIBS;

 public:
  static void gpu_variant(legate::TaskContext& context)
  {
    const auto proc = Processor::get_executing_processor();
    auto& lib       = get_cuda_libraries(proc);
    lib.finalize();
  }
};

static void __attribute__((constructor)) register_tasks(void)
{
  LoadCUDALibsTask::register_variants();
  UnloadCUDALibsTask::register_variants();
}

}  // namespace cunumeric
