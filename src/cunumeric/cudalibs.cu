/* Copyright 2021 NVIDIA Corporation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */

#include "legate.h"

#include "cudalibs.h"

#include <mutex>
#include <stdio.h>

namespace cunumeric {

using namespace Legion;

CUDALibraries::CUDALibraries() : cublas_(nullptr), cusolver_(nullptr) {}

CUDALibraries::~CUDALibraries() { finalize(); }

void CUDALibraries::finalize()
{
  if (cublas_ != nullptr) finalize_cublas();
  if (cusolver_ != nullptr) finalize_cusolver();
}

void CUDALibraries::finalize_cublas()
{
  hipblasStatus_t status = hipblasDestroy(cublas_);
  if (status != HIPBLAS_STATUS_SUCCESS) {
    fprintf(stderr,
            "Internal cuBLAS destruction failure "
            "with error code %d in cuNumeric\n",
            status);
    abort();
  }
  cublas_ = nullptr;
}

void CUDALibraries::finalize_cusolver()
{
  hipsolverStatus_t status = hipsolverDnDestroy(cusolver_);
  if (status != HIPSOLVER_STATUS_SUCCESS) {
    fprintf(stderr,
            "Internal cuSOLVER destruction failure "
            "with error code %d in cuNumeric\n",
            status);
    abort();
  }
  cusolver_ = nullptr;
}

hipblasHandle_t CUDALibraries::get_cublas()
{
  if (nullptr == cublas_) {
    hipblasStatus_t status = hipblasCreate(&cublas_);
    if (status != HIPBLAS_STATUS_SUCCESS) {
      fprintf(stderr,
              "Internal cuBLAS initialization failure "
              "with error code %d in cuNumeric\n",
              status);
      abort();
    }
    const char* disable_tensor_cores = getenv("CUNUMERIC_DISABLE_TENSOR_CORES");
    if (nullptr == disable_tensor_cores) {
      // No request to disable tensor cores so turn them on
      status = hipblasSetMathMode(cublas_, HIPBLAS_TENSOR_OP_MATH);
      if (status != HIPBLAS_STATUS_SUCCESS)
        fprintf(stderr, "WARNING: cuBLAS does not support Tensor cores!");
    }
  }
  return cublas_;
}

hipsolverHandle_t CUDALibraries::get_cusolver()
{
  if (nullptr == cusolver_) {
    hipsolverStatus_t status = hipsolverDnCreate(&cusolver_);
    if (status != HIPSOLVER_STATUS_SUCCESS) {
      fprintf(stderr,
              "Internal cuSOLVER initialization failure "
              "with error code %d in cuNumeric\n",
              status);
      abort();
    }
  }
  return cusolver_;
}

static CUDALibraries& get_cuda_libraries(Processor proc)
{
  if (proc.kind() != Processor::TOC_PROC) {
    fprintf(stderr, "Illegal request for CUDA libraries for non-GPU processor");
    LEGATE_ABORT
  }
  static std::mutex mut_cuda_libraries;
  static std::map<Processor, CUDALibraries> cuda_libraries;

  std::lock_guard<std::mutex> guard(mut_cuda_libraries);

  auto finder = cuda_libraries.find(proc);
  if (finder != cuda_libraries.end())
    return finder->second;
  else
    return cuda_libraries[proc];
}

cublasContext* get_cublas()
{
  const auto proc = Processor::get_executing_processor();
  auto& lib       = get_cuda_libraries(proc);
  return lib.get_cublas();
}

cusolverDnContext* get_cusolver()
{
  const auto proc = Processor::get_executing_processor();
  auto& lib       = get_cuda_libraries(proc);
  return lib.get_cusolver();
}

}  // namespace cunumeric
