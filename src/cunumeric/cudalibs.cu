/* Copyright 2021-2022 NVIDIA Corporation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */

#include "cunumeric.h"

#include "cudalibs.h"

#include <mutex>
#include <stdio.h>

namespace cunumeric {

using namespace Legion;

static Logger log_cudalibs("cunumeric.cudalibs");

cufftContext::cufftContext(cufftPlan* plan) : plan_(plan) {}

cufftContext::~cufftContext()
{
  auto hdl = handle();
  for (auto type : callback_types_) CHECK_CUFFT(hipfftXtClearCallback(hdl, type));
}

hipfftHandle cufftContext::handle() { return plan_->handle; }

size_t cufftContext::workarea_size() { return plan_->workarea; }

void cufftContext::set_callback(hipfftXtCallbackType type, void* callback, void* data)
{
  auto hdl = handle();
  if (callback_types_.find(type) != callback_types_.end())
    CHECK_CUFFT(hipfftXtClearCallback(hdl, type));
  void* callbacks[1] = {callback};
  void* datas[1]     = {data};
  CHECK_CUFFT(hipfftXtSetCallback(hdl, callbacks, type, datas));
  callback_types_.insert(type);
}

struct cufftPlanCache {
 private:
  // Maximum number of plans to keep per dimension
  static constexpr int32_t MAX_PLANS = 4;

 private:
  struct LRUEntry {
    std::unique_ptr<cufftPlan> plan{nullptr};
    DomainPoint fftshape{};
    uint32_t lru_index{0};
  };

 public:
  cufftPlanCache(hipfftType type);
  ~cufftPlanCache();

 public:
  cufftPlan* get_cufft_plan(const DomainPoint& size);

 private:
  using Cache = std::array<LRUEntry, MAX_PLANS>;
  std::array<Cache, LEGION_MAX_DIM + 1> cache_{};
  hipfftType type_;
};

cufftPlanCache::cufftPlanCache(hipfftType type) : type_(type)
{
  for (auto& cache : cache_)
    for (auto& entry : cache) assert(0 == entry.fftshape.dim);
}

cufftPlanCache::~cufftPlanCache()
{
  for (auto& cache : cache_)
    for (auto& entry : cache)
      if (entry.plan != nullptr) CHECK_CUFFT(hipfftDestroy(entry.plan->handle));
}

cufftPlan* cufftPlanCache::get_cufft_plan(const DomainPoint& size)
{
  int32_t match = -1;
  auto& cache   = cache_[size.dim];
  for (int32_t idx = 0; idx < MAX_PLANS; ++idx)
    if (cache[idx].fftshape == size) {
      match = idx;
      break;
    }

  cufftPlan* result{nullptr};
  // If there's no match, we create a new plan
  if (-1 == match) {
    log_cudalibs.debug() << "[cufftPlanCache] no match found for " << size << " (type: " << type_
                         << ")";
    int32_t plan_index = -1;
    for (int32_t idx = 0; idx < MAX_PLANS; ++idx) {
      auto& entry = cache[idx];
      if (nullptr == entry.plan) {
        log_cudalibs.debug() << "[cufftPlanCache] found empty entry " << idx << " (type: " << type_
                             << ")";
        entry.plan      = std::make_unique<cufftPlan>();
        entry.lru_index = idx;
        plan_index      = idx;
        break;
      } else if (entry.lru_index == MAX_PLANS - 1) {
        log_cudalibs.debug() << "[cufftPlanCache] evict entry " << idx << " for " << entry.fftshape
                             << " (type: " << type_ << ")";
        CHECK_CUFFT(hipfftDestroy(entry.plan->handle));
        plan_index = idx;
        break;
      }
    }
    assert(plan_index != -1);
    auto& entry    = cache[plan_index];
    entry.fftshape = size;
    result         = entry.plan.get();

    CHECK_CUFFT(hipfftCreate(&result->handle));
    CHECK_CUFFT(hipfftSetAutoAllocation(result->handle, 0 /*we'll do the allocation*/));

    std::vector<int32_t> n(size.dim);
    for (int32_t dim = 0; dim < size.dim; ++dim) n[dim] = size[dim];
    CHECK_CUFFT(hipfftMakePlanMany(result->handle,
                                  size.dim,
                                  n.data(),
                                  nullptr,
                                  1,
                                  1,
                                  nullptr,
                                  1,
                                  1,
                                  type_,
                                  1 /*batch*/,
                                  &result->workarea));
  }
  // Otherwise, we return the cached plan and adjust the LRU count
  else {
    log_cudalibs.debug() << "[cufftPlanCache] found match for " << size << " (type: " << type_
                         << ")";
    auto& entry = cache[match];
    result      = entry.plan.get();

    for (int32_t idx = 0; idx < MAX_PLANS; ++idx) {
      auto& other = cache[idx];
      if (other.lru_index < entry.lru_index) ++other.lru_index;
    }
    entry.lru_index = 0;
  }
  return result;
}

CUDALibraries::CUDALibraries()
  : finalized_(false), cublas_(nullptr), cusolver_(nullptr), cutensor_(nullptr), plan_caches_()
{
  CHECK_CUDA(hipStreamCreateWithFlags(&stream_, hipStreamNonBlocking));
}

CUDALibraries::~CUDALibraries() { finalize(); }

void CUDALibraries::finalize()
{
  if (finalized_) return;
  if (cublas_ != nullptr) finalize_cublas();
  if (cusolver_ != nullptr) finalize_cusolver();
  if (cutensor_ != nullptr) finalize_cutensor();
  for (auto& pair : plan_caches_) delete pair.second;
  hipStreamDestroy(stream_);
  finalized_ = true;
}

void CUDALibraries::finalize_cublas()
{
  CHECK_CUBLAS(hipblasDestroy(cublas_));
  cublas_ = nullptr;
}

void CUDALibraries::finalize_cusolver()
{
  CHECK_CUSOLVER(hipsolverDnDestroy(cusolver_));
  cusolver_ = nullptr;
}

void CUDALibraries::finalize_cutensor()
{
  delete cutensor_;
  cutensor_ = nullptr;
}

hipStream_t CUDALibraries::get_cached_stream() { return stream_; }

hipblasHandle_t CUDALibraries::get_cublas()
{
  if (nullptr == cublas_) {
    CHECK_CUBLAS(hipblasCreate(&cublas_));
    const char* disable_tensor_cores = getenv("CUNUMERIC_DISABLE_TENSOR_CORES");
    if (nullptr == disable_tensor_cores) {
      // No request to disable tensor cores so turn them on
      hipblasStatus_t status = hipblasSetMathMode(cublas_, HIPBLAS_TENSOR_OP_MATH);
      if (status != HIPBLAS_STATUS_SUCCESS)
        fprintf(stderr, "WARNING: cuBLAS does not support Tensor cores!");
    }
  }
  return cublas_;
}

hipsolverHandle_t CUDALibraries::get_cusolver()
{
  if (nullptr == cusolver_) CHECK_CUSOLVER(hipsolverDnCreate(&cusolver_));
  return cusolver_;
}

hiptensorHandle_t* CUDALibraries::get_cutensor()
{
  if (nullptr == cutensor_) {
    cutensor_ = new hiptensorHandle_t;
    CHECK_CUTENSOR(cutensorInit(cutensor_));
  }
  return cutensor_;
}

cufftContext CUDALibraries::get_cufft_plan(hipfftType type, const DomainPoint& size)
{
  auto finder = plan_caches_.find(type);
  cufftPlanCache* cache{nullptr};

  if (plan_caches_.end() == finder) {
    cache              = new cufftPlanCache(type);
    plan_caches_[type] = cache;
  } else
    cache = finder->second;
  return cufftContext(cache->get_cufft_plan(size));
}

static CUDALibraries& get_cuda_libraries(Processor proc)
{
  if (proc.kind() != Processor::TOC_PROC) {
    fprintf(stderr, "Illegal request for CUDA libraries for non-GPU processor");
    LEGATE_ABORT;
  }
  static std::mutex mut_cuda_libraries;
  static std::map<Processor, CUDALibraries> cuda_libraries;

  std::lock_guard<std::mutex> guard(mut_cuda_libraries);

  auto finder = cuda_libraries.find(proc);
  if (finder != cuda_libraries.end())
    return finder->second;
  else
    return cuda_libraries[proc];
}

hipStream_t get_cached_stream()
{
  const auto proc = Processor::get_executing_processor();
  auto& lib       = get_cuda_libraries(proc);
  return lib.get_cached_stream();
}

cublasContext* get_cublas()
{
  const auto proc = Processor::get_executing_processor();
  auto& lib       = get_cuda_libraries(proc);
  return lib.get_cublas();
}

cusolverDnContext* get_cusolver()
{
  const auto proc = Processor::get_executing_processor();
  auto& lib       = get_cuda_libraries(proc);
  return lib.get_cusolver();
}

hiptensorHandle_t* get_cutensor()
{
  const auto proc = Processor::get_executing_processor();
  auto& lib       = get_cuda_libraries(proc);
  return lib.get_cutensor();
}

cufftContext get_cufft_plan(hipfftType type, const Legion::DomainPoint& size)
{
  const auto proc = Processor::get_executing_processor();
  auto& lib       = get_cuda_libraries(proc);
  return lib.get_cufft_plan(type, size);
}

class LoadCUDALibsTask : public CuNumericTask<LoadCUDALibsTask> {
 public:
  static const int TASK_ID = CUNUMERIC_LOAD_CUDALIBS;

 public:
  static void gpu_variant(legate::TaskContext& context)
  {
    const auto proc = Processor::get_executing_processor();
    auto& lib       = get_cuda_libraries(proc);
    lib.get_cublas();
    lib.get_cusolver();
    lib.get_cutensor();
  }
};

class UnloadCUDALibsTask : public CuNumericTask<UnloadCUDALibsTask> {
 public:
  static const int TASK_ID = CUNUMERIC_UNLOAD_CUDALIBS;

 public:
  static void gpu_variant(legate::TaskContext& context)
  {
    const auto proc = Processor::get_executing_processor();
    auto& lib       = get_cuda_libraries(proc);
    lib.finalize();
  }
};

static void __attribute__((constructor)) register_tasks(void)
{
  LoadCUDALibsTask::register_variants();
  UnloadCUDALibsTask::register_variants();
}

}  // namespace cunumeric
