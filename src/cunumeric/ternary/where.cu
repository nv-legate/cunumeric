#include "hip/hip_runtime.h"
/* Copyright 2021-2022 NVIDIA Corporation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */

#include "cunumeric/ternary/where.h"
#include "cunumeric/ternary/where_template.inl"

#include "cunumeric/cuda_help.h"

namespace cunumeric {

template <typename VAL>
static __global__ void __launch_bounds__(THREADS_PER_BLOCK, MIN_CTAS_PER_SM)
  dense_kernel(size_t volume, VAL* out, const bool* mask, const VAL* in1, const VAL* in2)
{
  const size_t idx = global_tid_1d();
  if (idx >= volume) return;
  out[idx] = mask[idx] ? in1[idx] : in2[idx];
}

template <typename WriteAcc, typename MaskAcc, typename ReadAcc, typename Pitches, typename Rect>
static __global__ void __launch_bounds__(THREADS_PER_BLOCK, MIN_CTAS_PER_SM) generic_kernel(
  size_t volume, WriteAcc out, MaskAcc mask, ReadAcc in1, ReadAcc in2, Pitches pitches, Rect rect)
{
  const size_t idx = global_tid_1d();
  if (idx >= volume) return;
  auto point = pitches.unflatten(idx, rect.lo);
  out[point] = mask[point] ? in1[point] : in2[point];
}

template <Type CODE, int DIM>
struct WhereImplBody<VariantKind::GPU, CODE, DIM> {
  using VAL = legate_type_of<CODE>;

  void operator()(AccessorWO<VAL, DIM> out,
                  AccessorRO<bool, DIM> mask,
                  AccessorRO<VAL, DIM> in1,
                  AccessorRO<VAL, DIM> in2,
                  const Pitches<DIM - 1>& pitches,
                  const Rect<DIM>& rect,
                  bool dense) const
  {
    const size_t volume = rect.volume();
    const size_t blocks = (volume + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    auto stream         = get_cached_stream();
    if (dense) {
      size_t volume = rect.volume();
      auto outptr   = out.ptr(rect);
      auto maskptr  = mask.ptr(rect);
      auto in1ptr   = in1.ptr(rect);
      auto in2ptr   = in2.ptr(rect);
      dense_kernel<<<blocks, THREADS_PER_BLOCK, 0, stream>>>(
        volume, outptr, maskptr, in1ptr, in2ptr);
    } else {
      generic_kernel<<<blocks, THREADS_PER_BLOCK, 0, stream>>>(
        volume, out, mask, in1, in2, pitches, rect);
    }
    CHECK_CUDA_STREAM(stream);
  }
};

/*static*/ void WhereTask::gpu_variant(TaskContext& context)
{
  where_template<VariantKind::GPU>(context);
}

}  // namespace cunumeric
