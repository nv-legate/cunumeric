#include "hip/hip_runtime.h"
/* Copyright 2021-2022 NVIDIA Corporation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */

#include "cunumeric/matrix/trsm.h"
#include "cunumeric/matrix/trsm_template.inl"

#include "cunumeric/cuda_help.h"

namespace cunumeric {

using namespace legate;

template <typename Trsm, typename VAL>
static inline void trsm_template(
  Trsm trsm, VAL* lhs, const VAL* rhs, int32_t m, int32_t n, VAL alpha)
{
  auto context = get_cublas();
  auto stream  = get_cached_stream();
  CHECK_CUBLAS(hipblasSetStream(context, stream));

  // TODO: We need to expose these parameters to the API later we port scipy.linalg
  auto side   = HIPBLAS_SIDE_RIGHT;
  auto uplo   = HIPBLAS_FILL_MODE_LOWER;
  auto transa = HIPBLAS_OP_C;
  auto diag   = HIPBLAS_DIAG_NON_UNIT;

  CHECK_CUBLAS(trsm(context, side, uplo, transa, diag, m, n, &alpha, rhs, n, lhs, m));

  CHECK_CUDA_STREAM(stream);
}

template <>
struct TrsmImplBody<VariantKind::GPU, Type::FLOAT32> {
  void operator()(float* lhs, const float* rhs, int32_t m, int32_t n)
  {
    trsm_template(hipblasStrsm, lhs, rhs, m, n, 1.0F);
  }
};

template <>
struct TrsmImplBody<VariantKind::GPU, Type::FLOAT64> {
  void operator()(double* lhs, const double* rhs, int32_t m, int32_t n)
  {
    trsm_template(hipblasDtrsm, lhs, rhs, m, n, 1.0);
  }
};

template <>
struct TrsmImplBody<VariantKind::GPU, Type::COMPLEX64> {
  void operator()(complex<float>* lhs_, const complex<float>* rhs_, int32_t m, int32_t n)
  {
    auto lhs = reinterpret_cast<hipComplex*>(lhs_);
    auto rhs = reinterpret_cast<const hipComplex*>(rhs_);

    trsm_template(hipblasCtrsm, lhs, rhs, m, n, make_float2(1.0, 0.0));
  }
};

template <>
struct TrsmImplBody<VariantKind::GPU, Type::COMPLEX128> {
  void operator()(complex<double>* lhs_, const complex<double>* rhs_, int32_t m, int32_t n)
  {
    auto lhs = reinterpret_cast<hipDoubleComplex*>(lhs_);
    auto rhs = reinterpret_cast<const hipDoubleComplex*>(rhs_);

    trsm_template(hipblasZtrsm, lhs, rhs, m, n, make_double2(1.0, 0.0));
  }
};

/*static*/ void TrsmTask::gpu_variant(TaskContext& context)
{
  trsm_template<VariantKind::GPU>(context);
}

}  // namespace cunumeric
