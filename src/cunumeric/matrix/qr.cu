#include "hip/hip_runtime.h"
/* Copyright 2023 NVIDIA Corporation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */

#include "cunumeric/matrix/qr.h"
#include "cunumeric/matrix/qr_template.inl"

#include "cunumeric/cuda_help.h"
#include <vector>
namespace cunumeric {

using namespace legate;

template <typename GeqrfBufferSize,
          typename OrgqrBufferSize,
          typename Geqrf,
          typename Orgqr,
          typename VAL>
static inline void qr_template(GeqrfBufferSize geqrf_buffer_size,
                               OrgqrBufferSize orgqr_buffer_size,
                               Geqrf geqrf,
                               Orgqr orgqr,
                               int32_t m,
                               int32_t n,
                               int32_t k,
                               const VAL* a,
                               VAL* q,
                               VAL* r)
{
  auto handle = get_cusolver();
  auto stream = get_cached_stream();

  // m>=n : a[m][n], q[m][n] r[n][n]
  // m<n  : a[m][n], q[m][m] r[m][n]

  VAL* q_tmp = q;
  // if m < n:  q is not large enough to make compute inplace -> make tmp buffer
  if (m < n) {
    auto q_copy = create_buffer<VAL>(m * n, Memory::Kind::GPU_FB_MEM);
    q_tmp       = q_copy.ptr(0);
  }

  CHECK_CUDA(hipMemcpyAsync(q_tmp, a, sizeof(VAL) * m * n, hipMemcpyDeviceToDevice, stream));
  CHECK_CUDA(hipStreamSynchronize(stream));

  CHECK_CUSOLVER(hipsolverSetStream(handle, stream));

  auto tau  = create_buffer<VAL>(k, Memory::Kind::GPU_FB_MEM);
  auto info = create_buffer<int32_t>(1, Memory::Kind::Z_COPY_MEM);

  // compute and alloc buffer for geqrf
  int32_t lwork_geqrf, lwork_orgqr;
  CHECK_CUSOLVER(geqrf_buffer_size(handle, m, n, q_tmp, m, &lwork_geqrf));
  CHECK_CUSOLVER(orgqr_buffer_size(handle, m, n, k, q_tmp, m, tau.ptr(0), &lwork_orgqr));
  int32_t lwork_total = std::max(lwork_geqrf, lwork_orgqr);

  auto buffer = create_buffer<VAL>(lwork_total, Memory::Kind::GPU_FB_MEM);

  CHECK_CUSOLVER(
    geqrf(handle, m, n, q_tmp, m, tau.ptr(0), buffer.ptr(0), lwork_total, info.ptr(0)));
  CHECK_CUDA(hipStreamSynchronize(stream));

  if (info[0] != 0) throw legate::TaskException(QrTask::ERROR_MESSAGE);

  // extract R from upper triangular of geqrf result
  CHECK_CUDA(hipMemsetAsync(r, 0, k * n * sizeof(VAL), stream));
  for (int i = 0; i < k; ++i) {
    int elements = i + 1;
    if (i == k - 1 && n > k) elements = k * (n - k + 1);
    CHECK_CUDA(hipMemcpyAsync(
      r + i * k, q_tmp + i * m, sizeof(VAL) * elements, hipMemcpyDeviceToDevice, stream));
  }

  // assemble Q
  CHECK_CUSOLVER(
    orgqr(handle, m, k, k, q_tmp, m, tau.ptr(0), buffer.ptr(0), lwork_total, info.ptr(0)));
  CHECK_CUDA(hipStreamSynchronize(stream));

  if (info[0] != 0) throw legate::TaskException(QrTask::ERROR_MESSAGE);

  // if we used a tmp storage we still need to copy back Q
  if (q_tmp != q) {
    assert(n > m);
    CHECK_CUDA(hipMemcpyAsync(q, q_tmp, sizeof(VAL) * m * m, hipMemcpyDeviceToDevice, stream));
  }

  CHECK_CUDA_STREAM(stream);

#ifdef DEBUG_CUNUMERIC
  assert(info[0] == 0);
#endif
}

template <>
struct QrImplBody<VariantKind::GPU, Type::Code::FLOAT32> {
  void operator()(int32_t m, int32_t n, int32_t k, const float* a, float* q, float* r)
  {
    qr_template(hipsolverDnSgeqrf_bufferSize,
                hipsolverDnSorgqr_bufferSize,
                hipsolverDnSgeqrf,
                hipsolverDnSorgqr,
                m,
                n,
                k,
                a,
                q,
                r);
  }
};

template <>
struct QrImplBody<VariantKind::GPU, Type::Code::FLOAT64> {
  void operator()(int32_t m, int32_t n, int32_t k, const double* a, double* q, double* r)
  {
    qr_template(hipsolverDnDgeqrf_bufferSize,
                hipsolverDnDorgqr_bufferSize,
                hipsolverDnDgeqrf,
                hipsolverDnDorgqr,
                m,
                n,
                k,
                a,
                q,
                r);
  }
};

template <>
struct QrImplBody<VariantKind::GPU, Type::Code::COMPLEX64> {
  void operator()(
    int32_t m, int32_t n, int32_t k, const complex<float>* a, complex<float>* q, complex<float>* r)
  {
    qr_template(hipsolverDnCgeqrf_bufferSize,
                hipsolverDnCungqr_bufferSize,
                hipsolverDnCgeqrf,
                hipsolverDnCungqr,
                m,
                n,
                k,
                reinterpret_cast<const hipComplex*>(a),
                reinterpret_cast<hipComplex*>(q),
                reinterpret_cast<hipComplex*>(r));
  }
};

template <>
struct QrImplBody<VariantKind::GPU, Type::Code::COMPLEX128> {
  void operator()(int32_t m,
                  int32_t n,
                  int32_t k,
                  const complex<double>* a,
                  complex<double>* q,
                  complex<double>* r)
  {
    qr_template(hipsolverDnZgeqrf_bufferSize,
                hipsolverDnZungqr_bufferSize,
                hipsolverDnZgeqrf,
                hipsolverDnZungqr,
                m,
                n,
                k,
                reinterpret_cast<const hipDoubleComplex*>(a),
                reinterpret_cast<hipDoubleComplex*>(q),
                reinterpret_cast<hipDoubleComplex*>(r));
  }
};

/*static*/ void QrTask::gpu_variant(TaskContext& context)
{
  qr_template<VariantKind::GPU>(context);
}

}  // namespace cunumeric
