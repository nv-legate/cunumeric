/* Copyright 2021-2022 NVIDIA Corporation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */

#include "cunumeric/matrix/potrf.h"
#include "cunumeric/matrix/potrf_template.inl"

#include "cunumeric/cuda_help.h"

namespace cunumeric {

using namespace Legion;
using namespace legate;

template <typename PotrfBufferSize, typename Potrf, typename VAL>
static inline void potrf_template(
  PotrfBufferSize potrfBufferSize, Potrf potrf, VAL* array, int32_t m, int32_t n)
{
  auto uplo = HIPBLAS_FILL_MODE_LOWER;

  auto context = get_cusolver();
  auto stream  = get_cached_stream();
  CHECK_CUSOLVER(hipsolverSetStream(context, stream));

  int32_t bufferSize;
  CHECK_CUSOLVER(potrfBufferSize(context, uplo, n, array, m, &bufferSize));

  auto buffer = create_buffer<VAL>(bufferSize, Memory::Kind::GPU_FB_MEM);
  auto info   = create_buffer<int32_t>(1, Memory::Kind::Z_COPY_MEM);

  CHECK_CUSOLVER(potrf(context, uplo, n, array, m, buffer.ptr(0), bufferSize, info.ptr(0)));

  CHECK_CUDA_STREAM(stream);
}

template <>
struct PotrfImplBody<VariantKind::GPU, LegateTypeCode::FLOAT_LT> {
  void operator()(float* array, int32_t m, int32_t n)
  {
    potrf_template(hipsolverDnSpotrf_bufferSize, hipsolverDnSpotrf, array, m, n);
  }
};

template <>
struct PotrfImplBody<VariantKind::GPU, LegateTypeCode::DOUBLE_LT> {
  void operator()(double* array, int32_t m, int32_t n)
  {
    potrf_template(hipsolverDnDpotrf_bufferSize, hipsolverDnDpotrf, array, m, n);
  }
};

template <>
struct PotrfImplBody<VariantKind::GPU, LegateTypeCode::COMPLEX64_LT> {
  void operator()(complex<float>* array, int32_t m, int32_t n)
  {
    potrf_template(
      hipsolverDnCpotrf_bufferSize, hipsolverDnCpotrf, reinterpret_cast<hipComplex*>(array), m, n);
  }
};

template <>
struct PotrfImplBody<VariantKind::GPU, LegateTypeCode::COMPLEX128_LT> {
  void operator()(complex<double>* array, int32_t m, int32_t n)
  {
    potrf_template(hipsolverDnZpotrf_bufferSize,
                   hipsolverDnZpotrf,
                   reinterpret_cast<hipDoubleComplex*>(array),
                   m,
                   n);
  }
};

/*static*/ void PotrfTask::gpu_variant(TaskContext& context)
{
  potrf_template<VariantKind::GPU>(context);
}

}  // namespace cunumeric
