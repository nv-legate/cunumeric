#include "hip/hip_runtime.h"
/* Copyright 2021-2022 NVIDIA Corporation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */

#include "cunumeric/matrix/trilu.h"
#include "cunumeric/matrix/trilu_template.inl"

#include "cunumeric/cuda_help.h"

namespace cunumeric {

using namespace legate;

template <typename VAL, int32_t DIM, bool LOWER, bool C_ORDER>
static __global__ void __launch_bounds__(THREADS_PER_BLOCK, MIN_CTAS_PER_SM)
  trilu_kernel(AccessorWO<VAL, DIM> out,
               AccessorRO<VAL, DIM> in,
               Pitches<DIM - 1, C_ORDER> pitches,
               Point<DIM> lo,
               size_t volume,
               int32_t k)
{
  const size_t idx = global_tid_1d();
  if (idx >= volume) return;

  if (LOWER) {
    auto p = pitches.unflatten(idx, lo);
    if (p[DIM - 2] + k >= p[DIM - 1])
      out[p] = in[p];
    else
      out[p] = 0;
  } else {
    auto p = pitches.unflatten(idx, lo);
    if (p[DIM - 2] + k <= p[DIM - 1])
      out[p] = in[p];
    else
      out[p] = 0;
  }
}

template <LegateTypeCode CODE, int32_t DIM, bool LOWER>
struct TriluImplBody<VariantKind::GPU, CODE, DIM, LOWER> {
  using VAL = legate_type_of<CODE>;

  template <bool C_ORDER>
  void operator()(const AccessorWO<VAL, DIM>& out,
                  const AccessorRO<VAL, DIM>& in,
                  const Pitches<DIM - 1, C_ORDER>& pitches,
                  const Point<DIM>& lo,
                  size_t volume,
                  int32_t k) const
  {
    const size_t blocks = (volume + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    auto stream         = get_cached_stream();
    trilu_kernel<VAL, DIM, LOWER, C_ORDER>
      <<<blocks, THREADS_PER_BLOCK, 0, stream>>>(out, in, pitches, lo, volume, k);
    CHECK_CUDA_STREAM(stream);
  }
};

/*static*/ void TriluTask::gpu_variant(TaskContext& context)
{
  trilu_template<VariantKind::GPU>(context);
}

}  // namespace cunumeric
