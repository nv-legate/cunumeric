#include "hip/hip_runtime.h"
/* Copyright 2021-2022 NVIDIA Corporation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */

#include "cunumeric/matrix/transpose.h"
#include "cunumeric/matrix/transpose_template.inl"

#include "cunumeric/cuda_help.h"

namespace cunumeric {

#define TILE_DIM 32
#define BLOCK_ROWS 8

template <typename VAL>
__global__ static void __launch_bounds__((TILE_DIM * BLOCK_ROWS), MIN_CTAS_PER_SM)
  transpose_2d_logical(const AccessorWO<VAL, 2> out,
                       const AccessorRO<VAL, 2> in,
                       const Point<2> lo_in,
                       const Point<2> hi_in,
                       const Point<2> lo_out,
                       const Point<2> hi_out)
{
  __shared__ VAL tile[TILE_DIM][TILE_DIM + 1 /*avoid bank conflicts*/];

  // These are reversed here for coalescing
  coord_t x = blockIdx.y * TILE_DIM + threadIdx.y;
  coord_t y = blockIdx.x * TILE_DIM + threadIdx.x;

  // Check to see if we hit our y-bounds, if so we can just mask off those threads
  if ((lo_in[1] + y) <= hi_in[1]) {
    // Check to see if we're going to hit our x-bounds while striding
    if ((lo_in[0] + (blockIdx.y + 1) * TILE_DIM - 1) <= hi_in[0]) {
// No overflow case
#pragma unroll
      for (int i = 0; i < TILE_DIM; i += BLOCK_ROWS)
        tile[threadIdx.y + i][threadIdx.x] = in[lo_in + Point<2>(x + i, y)];
    } else {
// Overflow case
#pragma unroll
      for (int i = 0; i < TILE_DIM; i += BLOCK_ROWS)
        if ((lo_in[0] + x + i) <= hi_in[0])
          tile[threadIdx.y + i][threadIdx.x] = in[lo_in + Point<2>(x + i, y)];
    }
  }
  // Make sure all the data is in shared memory
  __syncthreads();

  // Transpose the coordinates
  x = blockIdx.x * TILE_DIM + threadIdx.y;
  y = blockIdx.y * TILE_DIM + threadIdx.x;

  // Check to see if we hit our y-bounds, if so we can just mask off those threads
  if ((lo_out[1] + y) <= hi_out[1]) {
    // Check to see if we're going to hit our x-bounds while striding
    if ((lo_out[0] + (blockIdx.x + 1) * TILE_DIM - 1) <= hi_out[0]) {
// No overflow case
#pragma unroll
      for (int i = 0; i < TILE_DIM; i += BLOCK_ROWS)
        out[lo_out + Point<2>(x + i, y)] = tile[threadIdx.x][threadIdx.y + i];
    } else {
// Overflow case
#pragma unroll
      for (int i = 0; i < TILE_DIM; i += BLOCK_ROWS)
        if ((lo_out[0] + x + i) <= hi_out[0])
          out[lo_out + Point<2>(x + i, y)] = tile[threadIdx.x][threadIdx.y + i];
    }
  }
}

template <typename VAL>
__global__ static void __launch_bounds__((TILE_DIM * BLOCK_ROWS), MIN_CTAS_PER_SM)
  transpose_2d_physical(const AccessorWO<VAL, 2> out,
                        const AccessorRO<VAL, 2> in,
                        const Point<2> lo_in,
                        const Point<2> hi_in,
                        const Point<2> lo_out,
                        const Point<2> hi_out)
{
  __shared__ VAL tile[TILE_DIM][TILE_DIM + 1 /*avoid bank conflicts*/];

  // These are reversed here for coalescing
  coord_t x = blockIdx.y * TILE_DIM + threadIdx.y;
  coord_t y = blockIdx.x * TILE_DIM + threadIdx.x;

  // Check to see if we hit our y-bounds, if so we can just mask off those threads
  if ((lo_in[1] + y) <= hi_in[1]) {
    // Check to see if we're going to hit our x-bounds while striding
    if ((lo_in[0] + (blockIdx.y + 1) * TILE_DIM - 1) <= hi_in[0]) {
// No overflow case
#pragma unroll
      for (int i = 0; i < TILE_DIM; i += BLOCK_ROWS)
        tile[threadIdx.y + i][threadIdx.x] = in[lo_in + Point<2>(x + i, y)];
    } else {
// Overflow case
#pragma unroll
      for (int i = 0; i < TILE_DIM; i += BLOCK_ROWS)
        if ((lo_in[0] + x + i) <= hi_in[0])
          tile[threadIdx.y + i][threadIdx.x] = in[lo_in + Point<2>(x + i, y)];
    }
  }

  // Make sure all the data is in shared memory
  __syncthreads();

  x = blockIdx.y * TILE_DIM + threadIdx.x;
  y = blockIdx.x * TILE_DIM + threadIdx.y;

  // Check to see if we hit our x-bounds, if so we can just mask off those threads
  if ((lo_out[0] + x) <= hi_out[0]) {
    // Check to see if we're going to hit our y-bounds while striding
    if ((lo_out[1] + (blockIdx.x + 1) * TILE_DIM - 1) <= hi_out[1]) {
// No overflow case
#pragma unroll
      for (int i = 0; i < TILE_DIM; i += BLOCK_ROWS)
        out[lo_out + Point<2>(x, y + i)] = tile[threadIdx.x][threadIdx.y + i];
    } else {
// Overflow case
#pragma unroll
      for (int i = 0; i < TILE_DIM; i += BLOCK_ROWS)
        if ((lo_out[1] + y + i) <= hi_out[1])
          out[lo_out + Point<2>(x, y + i)] = tile[threadIdx.x][threadIdx.y + i];
    }
  }
}

template <LegateTypeCode CODE>
struct TransposeImplBody<VariantKind::GPU, CODE> {
  using VAL = legate_type_of<CODE>;

  void operator()(const Rect<2>& out_rect,
                  const Rect<2>& in_rect,
                  const AccessorWO<VAL, 2>& out,
                  const AccessorRO<VAL, 2>& in,
                  bool logical) const
  {
    const coord_t m = (in_rect.hi[0] - in_rect.lo[0]) + 1;
    const coord_t n = (in_rect.hi[1] - in_rect.lo[1]) + 1;
    const dim3 blocks((n + TILE_DIM - 1) / TILE_DIM, (m + TILE_DIM - 1) / TILE_DIM, 1);
    const dim3 threads(TILE_DIM, BLOCK_ROWS, 1);

    auto stream = get_cached_stream();
    if (logical)
      transpose_2d_logical<VAL>
        <<<blocks, threads, 0, stream>>>(out, in, in_rect.lo, in_rect.hi, out_rect.lo, out_rect.hi);
    else
      transpose_2d_physical<VAL>
        <<<blocks, threads, 0, stream>>>(out, in, in_rect.lo, in_rect.hi, out_rect.lo, out_rect.hi);
    CHECK_CUDA_STREAM(stream);
  }
};

/*static*/ void TransposeTask::gpu_variant(TaskContext& context)
{
  transpose_template<VariantKind::GPU>(context);
}

}  // namespace cunumeric
