/* Copyright 2022 NVIDIA Corporation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */

#include "cunumeric/sort/cub_sort.cuh"

namespace cunumeric {

void cub_local_sort(const int32_t* values_in,
                    int32_t* values_out,
                    const int64_t* indices_in,
                    int64_t* indices_out,
                    const size_t volume,
                    const size_t sort_dim_size,
                    hipStream_t stream)
{
  detail::cub_local_sort(
    values_in, values_out, indices_in, indices_out, volume, sort_dim_size, stream);
}

}  // namespace cunumeric
