/* Copyright 2022 NVIDIA Corporation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */

#include "cunumeric/sort/thrust_sort.cuh"

namespace cunumeric {

void thrust_local_sort(const complex<double>* values_in,
                       complex<double>* values_out,
                       const int64_t* indices_in,
                       int64_t* indices_out,
                       const size_t volume,
                       const size_t sort_dim_size,
                       const bool stable,
                       hipStream_t stream)
{
  detail::thrust_local_sort(
    values_in, values_out, indices_in, indices_out, volume, sort_dim_size, stable, stream);
}

}  // namespace cunumeric
