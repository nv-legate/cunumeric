#include "hip/hip_runtime.h"
/* Copyright 2021-2022 NVIDIA Corporation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */

#include "cunumeric/unary/unary_op.h"
#include "cunumeric/unary/unary_op_template.inl"

#include "cunumeric/cuda_help.h"

namespace cunumeric {

using namespace Legion;

template <typename Function, typename ARG, typename RES>
static __global__ void __launch_bounds__(THREADS_PER_BLOCK, MIN_CTAS_PER_SM)
  dense_kernel(size_t volume, Function func, RES* out, const ARG* in)
{
  const size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= volume) return;
  out[idx] = func(in[idx]);
}

template <typename Function, typename ReadAcc, typename WriteAcc, typename Pitches, typename Rect>
static __global__ void __launch_bounds__(THREADS_PER_BLOCK, MIN_CTAS_PER_SM)
  generic_kernel(size_t volume, Function func, WriteAcc out, ReadAcc in, Pitches pitches, Rect rect)
{
  const size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= volume) return;
  auto point = pitches.unflatten(idx, rect.lo);
  out[point] = func(in[point]);
}

template <UnaryOpCode OP_CODE, LegateTypeCode CODE, int DIM>
struct UnaryOpImplBody<VariantKind::GPU, OP_CODE, CODE, DIM> {
  using OP  = UnaryOp<OP_CODE, CODE>;
  using ARG = typename OP::T;
  using RES = std::result_of_t<OP(ARG)>;

  void operator()(OP func,
                  AccessorWO<RES, DIM> out,
                  AccessorRO<ARG, DIM> in,
                  const Pitches<DIM - 1>& pitches,
                  const Rect<DIM>& rect,
                  bool dense) const
  {
    const size_t volume = rect.volume();
    const size_t blocks = (volume + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    auto stream         = get_cached_stream();
    if (dense) {
      auto outptr = out.ptr(rect);
      auto inptr  = in.ptr(rect);
      dense_kernel<<<blocks, THREADS_PER_BLOCK, 0, stream>>>(volume, func, outptr, inptr);
    } else {
      generic_kernel<<<blocks, THREADS_PER_BLOCK, 0, stream>>>(
        volume, func, out, in, pitches, rect);
    }
    CHECK_CUDA_STREAM(stream);
  }
};

template <typename Function, typename LHS, typename RHS1, typename RHS2>
static __global__ void __launch_bounds__(THREADS_PER_BLOCK, MIN_CTAS_PER_SM)
  dense_kernel_multiout(size_t volume, Function func, LHS* lhs, const RHS1* rhs1, RHS2* rhs2)
{
  const size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= volume) return;
  lhs[idx] = func(rhs1[idx], &rhs2[idx]);
}

template <typename Function,
          typename LHSAcc,
          typename RHS1Acc,
          typename RHS2Acc,
          typename Pitches,
          typename Rect>
static __global__ void __launch_bounds__(THREADS_PER_BLOCK, MIN_CTAS_PER_SM)
  generic_kernel_multiout(size_t volume,
                          Function func,
                          LHSAcc lhs,
                          RHS1Acc rhs1,
                          RHS2Acc rhs2,
                          Pitches pitches,
                          Rect rect)
{
  const size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= volume) return;
  auto point = pitches.unflatten(idx, rect.lo);
  lhs[point] = func(rhs1[point], rhs2.ptr(point));
}

template <UnaryOpCode OP_CODE, LegateTypeCode CODE, int DIM>
struct MultiOutUnaryOpImplBody<VariantKind::GPU, OP_CODE, CODE, DIM> {
  using OP   = MultiOutUnaryOp<OP_CODE, CODE>;
  using RHS1 = typename OP::RHS1;
  using RHS2 = typename OP::RHS2;
  using LHS  = std::result_of_t<OP(RHS1, RHS2*)>;

  void operator()(OP func,
                  AccessorWO<LHS, DIM> lhs,
                  AccessorRO<RHS1, DIM> rhs1,
                  AccessorWO<RHS2, DIM> rhs2,
                  const Pitches<DIM - 1>& pitches,
                  const Rect<DIM>& rect,
                  bool dense) const
  {
    const size_t volume = rect.volume();
    const size_t blocks = (volume + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    auto stream         = get_cached_stream();
    if (dense) {
      auto lhsptr  = lhs.ptr(rect);
      auto rhs1ptr = rhs1.ptr(rect);
      auto rhs2ptr = rhs2.ptr(rect);
      dense_kernel_multiout<<<blocks, THREADS_PER_BLOCK, 0, stream>>>(
        volume, func, lhsptr, rhs1ptr, rhs2ptr);
    } else {
      generic_kernel_multiout<<<blocks, THREADS_PER_BLOCK, 0, stream>>>(
        volume, func, lhs, rhs1, rhs2, pitches, rect);
    }
    CHECK_CUDA_STREAM(stream);
  }
};

/*static*/ void UnaryOpTask::gpu_variant(TaskContext& context)
{
  unary_op_template<VariantKind::GPU>(context);
}

}  // namespace cunumeric
