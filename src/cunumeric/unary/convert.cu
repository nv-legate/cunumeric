#include "hip/hip_runtime.h"
/* Copyright 2021-2022 NVIDIA Corporation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */

#include "cunumeric/unary/convert.h"
#include "cunumeric/unary/convert_template.inl"

#include "cunumeric/cuda_help.h"

namespace cunumeric {

template <typename Function, typename ARG, typename RES>
static __global__ void __launch_bounds__(THREADS_PER_BLOCK, MIN_CTAS_PER_SM)
  dense_kernel(size_t volume, Function func, RES* out, const ARG* in)
{
  const size_t idx = global_tid_1d();
  if (idx >= volume) return;
  out[idx] = func(in[idx]);
}

template <typename Function, typename ReadAcc, typename WriteAcc, typename Pitches, typename Rect>
static __global__ void __launch_bounds__(THREADS_PER_BLOCK, MIN_CTAS_PER_SM)
  generic_kernel(size_t volume, Function func, WriteAcc out, ReadAcc in, Pitches pitches, Rect rect)
{
  const size_t idx = global_tid_1d();
  if (idx >= volume) return;
  auto point = pitches.unflatten(idx, rect.lo);
  out[point] = func(in[point]);
}

template <ConvertCode NAN_OP, Type DST_TYPE, Type SRC_TYPE, int DIM>
struct ConvertImplBody<VariantKind::GPU, NAN_OP, DST_TYPE, SRC_TYPE, DIM> {
  using OP  = ConvertOp<NAN_OP, DST_TYPE, SRC_TYPE>;
  using SRC = legate_type_of<SRC_TYPE>;
  using DST = legate_type_of<DST_TYPE>;

  void operator()(OP func,
                  AccessorWO<DST, DIM> out,
                  AccessorRO<SRC, DIM> in,
                  const Pitches<DIM - 1>& pitches,
                  const Rect<DIM>& rect,
                  bool dense) const
  {
    const size_t volume = rect.volume();
    const size_t blocks = (volume + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    auto stream         = get_cached_stream();
    if (dense) {
      auto outptr = out.ptr(rect);
      auto inptr  = in.ptr(rect);
      dense_kernel<<<blocks, THREADS_PER_BLOCK, 0, stream>>>(volume, func, outptr, inptr);
    } else {
      generic_kernel<<<blocks, THREADS_PER_BLOCK, 0, stream>>>(
        volume, func, out, in, pitches, rect);
    }
    CHECK_CUDA_STREAM(stream);
  }
};

/*static*/ void ConvertTask::gpu_variant(TaskContext& context)
{
  convert_template<VariantKind::GPU>(context);
}

}  // namespace cunumeric
